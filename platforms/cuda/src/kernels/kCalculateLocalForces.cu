#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"

extern __shared__ Vectors sV[];
static __constant__ cudaGmxSimulation cSim;

/* Cuda compiler on Windows does not recognized "static const float" values */
#define LOCAL_HACK_PI 3.1415926535897932384626433832795

#define DOT3(v1, v2) (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z)

#define GETNORMEDDOTPRODUCT(v1, v2, dp) \
{ \
    dp          = DOT3(v1, v2); \
    float norm1 = DOT3(v1, v1); \
    float norm2 = DOT3(v2, v2); \
    dp /= sqrtf(norm1 * norm2); \
    dp = min(dp, 1.0f); \
    dp = max(dp, -1.0f); \
}

#define CROSS_PRODUCT(v1, v2, c) \
    c.x = v1.y * v2.z - v1.z * v2.y; \
    c.y = v1.z * v2.x - v1.x * v2.z; \
    c.z = v1.x * v2.y - v1.y * v2.x;

#define GETPREFACTORSGIVENANGLECOSINE(cosine, param, dEdR) \
{ \
   float angle          = acosf(cosine); \
   float deltaIdeal     = angle - (param.x * (LOCAL_HACK_PI / 180.0f)); \
   dEdR                 = param.y * deltaIdeal; \
}

#define GETENERGYGIVENANGLECOSINE(cosine, param, dEdR) \
{ \
   float angle          = acosf(cosine); \
   float deltaIdeal     = angle - (param.x * (LOCAL_HACK_PI / 180.0f)); \
   dEdR                 = param.y * deltaIdeal * deltaIdeal; \
}

#define GETANGLEBETWEENTWOVECTORS(v1, v2, angle) \
{ \
    float dp; \
    GETNORMEDDOTPRODUCT(v1, v2, dp); \
    if (dp > 0.99f || dp < -0.99f) { \
        float4 cross; \
        CROSS_PRODUCT(v1, v2, cross); \
        float scale = DOT3(v1, v1)*DOT3(v2, v2); \
        angle = asinf(sqrtf(DOT3(cross, cross)/scale)); \
        if (dp < 0.0f) \
            angle = LOCAL_HACK_PI-angle; \
    } \
    else { \
        angle = acosf(dp); \
    } \
}

#define GETDIHEDRALANGLEBETWEENTHREEVECTORS(vector1, vector2, vector3, signVector, cp0, cp1, angle) \
{ \
    CROSS_PRODUCT(vector1, vector2, cp0); \
    CROSS_PRODUCT(vector2, vector3, cp1); \
    GETANGLEBETWEENTWOVECTORS(cp0, cp1, angle); \
    float dp = DOT3(signVector, cp1); \
    angle = (dp >= 0) ? angle : -angle; \
}                                                          

#define GETDIHEDRALANGLECOSINEBETWEENTHREEVECTORS(vector1, vector2, vector3, signVector, cp0, cp1, angle, cosine) \
{ \
    CROSS_PRODUCT(vector1, vector2, cp0); \
    CROSS_PRODUCT(vector2, vector3, cp1); \
    GETANGLEBETWEENTWOVECTORS(cp0, cp1, angle); \
    float dp = DOT3(signVector, cp1); \
    angle = (dp >= 0) ? angle : -angle; \
    cosine = cosf(angle); \
}

void SetCalculateLocalForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateLocalForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}
    

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_LOCALFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
void kCalculateLocalForces_kernel()
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    Vectors* A = &sV[threadIdx.x];

    float energy = 0.0f;

    while (pos < cSim.bond_offset)
    {
        if (pos < cSim.bonds)
        {
            int4   atom         = cSim.pBondID[pos];
            float4 atomA        = cSim.pPosq[atom.x];
            float4 atomB        = cSim.pPosq[atom.y];
            float2 bond         = cSim.pBondParameter[pos];
            float dx            = atomB.x - atomA.x;
            float dy            = atomB.y - atomA.y;
            float dz            = atomB.z - atomA.z;
            float r2            = dx * dx + dy * dy + dz * dz;
            float r             = sqrtf(r2);
            float deltaIdeal    = r - bond.x;
/* E */     energy             += 0.5f * bond.y * deltaIdeal * deltaIdeal;
            float dEdR          = bond.y * deltaIdeal;
            dEdR                = (r > 0.0f) ? (dEdR / r) : 0.0f;
//            printf("D: %11.4f %11.4f %11.4f %11.4f %11.4f %11.4f\n", dx, dy, dz, r, deltaIdeal, dEdR);
            dx                 *= dEdR;
            dy                 *= dEdR;
            dz                 *= dEdR;
            unsigned int offsetA                = atom.x + atom.z * cSim.stride;
            unsigned int offsetB                = atom.y + atom.w * cSim.stride;
            float4 forceA                       = cSim.pForce4[offsetA];
            float4 forceB                       = cSim.pForce4[offsetB];
            forceA.x                           += dx;
            forceA.y                           += dy;
            forceA.z                           += dz;
            forceB.x                           -= dx;
            forceB.y                           -= dy;
            forceB.z                           -= dz;
            cSim.pForce4[offsetA]               = forceA;
            cSim.pForce4[offsetB]               = forceB;    
        }
        pos += blockDim.x * gridDim.x;
    }

    while (pos < cSim.bond_angle_offset)
    {
        unsigned int pos1   = pos - cSim.bond_offset;
        if (pos1 < cSim.bond_angles)
        {
            int4   atom1            = cSim.pBondAngleID1[pos1];  
            float2 bond_angle       = cSim.pBondAngleParameter[pos1];
            float4 a1               = cSim.pPosq[atom1.x];
            float4 a2               = cSim.pPosq[atom1.y];
            float4 a3               = cSim.pPosq[atom1.z];
            A->v0.x                 = a2.x - a1.x;
            A->v0.y                 = a2.y - a1.y;
            A->v0.z                 = a2.z - a1.z;
            A->v1.x                 = a2.x - a3.x;
            A->v1.y                 = a2.y - a3.y;
            A->v1.z                 = a2.z - a3.z;
            float3 cp;
            CROSS_PRODUCT(A->v0, A->v1, cp);
            float rp                = DOT3(cp, cp); //cx * cx + cy * cy + cz * cz;
            rp                      = max(sqrtf(rp), 1.0e-06f);
            float r21               = DOT3(A->v0, A->v0); // dx1 * dx1 + dy1 * dy1 + dz1 * dz1;
            float r23               = DOT3(A->v1, A->v1); // dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
            float dot               = DOT3(A->v0, A->v1); // dx1 * dx2 + dy1 * dy2 + dz1 * dz2;
            float cosine            = max(-1.0f, min(1.0f, dot / sqrtf(r21 * r23)));

            float angle_energy;
/* E */     GETENERGYGIVENANGLECOSINE(cosine, bond_angle, angle_energy);
            energy                 += 0.5f*angle_energy;

            float dEdR;
            GETPREFACTORSGIVENANGLECOSINE(cosine, bond_angle, dEdR);
            //printf("%11.4f %11.4f\n", cosine, dEdR);
            float termA             =  dEdR / (r21 * rp);
            float termC             = -dEdR / (r23 * rp);
            float3 c21;
            float3 c23;
            CROSS_PRODUCT(A->v0, cp, c21);
            CROSS_PRODUCT(A->v1, cp, c23);
            c21.x                  *= termA;
            c21.y                  *= termA;
            c21.z                  *= termA;
            c23.x                  *= termC;
            c23.y                  *= termC;
            c23.z                  *= termC;
            int2 atom2              = cSim.pBondAngleID2[pos1];
            unsigned int offset     = atom1.x + atom1.w * cSim.stride;
            float4 force            = cSim.pForce4[offset]; 
            force.x                += c21.x;
            force.y                += c21.y;
            force.z                += c21.z;
            cSim.pForce4[offset]    = force;
            offset                  = atom1.y + atom2.x * cSim.stride;
            force                   = cSim.pForce4[offset];
            force.x                -= (c21.x + c23.x);
            force.y                -= (c21.y + c23.y);
            force.z                -= (c21.z + c23.z);
            cSim.pForce4[offset]    = force;
            offset                  = atom1.z + atom2.y * cSim.stride;
            force                   = cSim.pForce4[offset];
            force.x                += c23.x;
            force.y                += c23.y;
            force.z                += c23.z;
            cSim.pForce4[offset]    = force;
        }
        pos += blockDim.x * gridDim.x;
    }

    while (pos < cSim.dihedral_offset)
    {
        unsigned int pos1 = pos - cSim.bond_angle_offset;
        if (pos1 < cSim.dihedrals)
        {
            int4   atom1        = cSim.pDihedralID1[pos1];  
            float4 atomA        = cSim.pPosq[atom1.x];
            float4 atomB        = cSim.pPosq[atom1.y];
            float4 atomC        = cSim.pPosq[atom1.z];
            float4 atomD        = cSim.pPosq[atom1.w];            
            A->v0.x             = atomA.x - atomB.x;
            A->v0.y             = atomA.y - atomB.y;
            A->v0.z             = atomA.z - atomB.z;
            A->v1.x             = atomC.x - atomB.x;
            A->v1.y             = atomC.y - atomB.y;
            A->v1.z             = atomC.z - atomB.z;
            A->v2.x             = atomC.x - atomD.x;
            A->v2.y             = atomC.y - atomD.y;
            A->v2.z             = atomC.z - atomD.z; 
            float3 cp0, cp1;
            float dihedralAngle;
            GETDIHEDRALANGLEBETWEENTHREEVECTORS(A->v0, A->v1, A->v2, A->v0, cp0, cp1, dihedralAngle);
            float4 dihedral         = cSim.pDihedralParameter[pos1];
            float deltaAngle        = dihedral.z * dihedralAngle - (dihedral.y * LOCAL_HACK_PI / 180.0f);

	    // ATTENTION: This section leads to a divergent deltaAngle values wrt
	    // forces and energies. We separate the case dihedral.z = n = 0, which
	    // is treated by the calculation of energies via a harmonic potential
/* E */     if (dihedral.z) energy += dihedral.x * (1.0f + cosf(deltaAngle));
/* E */     else
	    {
		float deltaAngle    = dihedralAngle - dihedral.y;
		if (deltaAngle < -LOCAL_HACK_PI) deltaAngle += 2.0f * LOCAL_HACK_PI;
		else if (deltaAngle > LOCAL_HACK_PI) deltaAngle -= 2.0f * LOCAL_HACK_PI;
                energy             += dihedral.x * deltaAngle * deltaAngle;
	    }

            float sinDeltaAngle     = sinf(deltaAngle);
            float dEdAngle          = -dihedral.x * dihedral.z * sinDeltaAngle;
            float normCross1        = DOT3(cp0, cp0);
            float normBC            = sqrtf(DOT3(A->v1, A->v1));
            float4 ff;
            ff.x                    = (-dEdAngle * normBC) / normCross1;
            float normCross2        = DOT3(cp1, cp1);
            ff.w                    = (dEdAngle * normBC) / normCross2;
            float dp                = 1.0f / DOT3(A->v1, A->v1);
            ff.y                    = DOT3(A->v0, A->v1) * dp;
            ff.z                    = DOT3(A->v2, A->v1) * dp;
            int4  atom2             = cSim.pDihedralID2[pos1];   
            float3 internalF0;
            float3 internalF3;
            float3 s;
            
//            printf("%4d: %9.4f %9.4f %9.4f %9.4f\n", pos1, ff.x, ff.y, ff.z, ff.w);  
            unsigned int offset                 = atom1.x + atom2.x * cSim.stride;
            float4 force                        = cSim.pForce4[offset]; 
            internalF0.x                        = ff.x * cp0.x; 
            force.x                            += internalF0.x;
            internalF0.y                        = ff.x * cp0.y;
            force.y                            += internalF0.y;
            internalF0.z                        = ff.x * cp0.z;       
            force.z                            += internalF0.z;
            cSim.pForce4[offset]                = force;
            
            //printf("%4d - 0: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.w + atom2.w * cSim.stride;
            force                               = cSim.pForce4[offset];
            internalF3.x                        = ff.w * cp1.x;
            force.x                            += internalF3.x;
            internalF3.y                        = ff.w * cp1.y;
            force.y                            += internalF3.y;
            internalF3.z                        = ff.w * cp1.z;
            force.z                            += internalF3.z;
            cSim.pForce4[offset]                = force;
            
           // printf("%4d - 3: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            s.x                                 = ff.y * internalF0.x - ff.z * internalF3.x;   
            s.y                                 = ff.y * internalF0.y - ff.z * internalF3.y;  
            s.z                                 = ff.y * internalF0.z - ff.z * internalF3.z;        
            offset                              = atom1.y + atom2.y * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF0.x + s.x;
            force.y                            += -internalF0.y + s.y;
            force.z                            += -internalF0.z + s.z;
            cSim.pForce4[offset]                = force;
            
            //printf("%4d - 1: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.z + atom2.z * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF3.x - s.x;
            force.y                            += -internalF3.y - s.y;
            force.z                            += -internalF3.z - s.z;
            cSim.pForce4[offset]                = force;
            //printf("%4d - 2: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
        }
        pos += blockDim.x * gridDim.x;
    }

    // Ryckaert Bellemans dihedrals
    while (pos < cSim.rb_dihedral_offset)
    {
        unsigned int pos1 = pos - cSim.dihedral_offset;
        if (pos1 < cSim.rb_dihedrals)
        {
            int4   atom1        = cSim.pRbDihedralID1[pos1];
            float4 atomA        = cSim.pPosq[atom1.x];
            float4 atomB        = cSim.pPosq[atom1.y];
            float4 atomC        = cSim.pPosq[atom1.z];
            float4 atomD        = cSim.pPosq[atom1.w];
            A->v0.x             = atomA.x - atomB.x;
            A->v0.y             = atomA.y - atomB.y;
            A->v0.z             = atomA.z - atomB.z;
            A->v1.x             = atomC.x - atomB.x;
            A->v1.y             = atomC.y - atomB.y;
            A->v1.z             = atomC.z - atomB.z;
            A->v2.x             = atomC.x - atomD.x;
            A->v2.y             = atomC.y - atomD.y;
            A->v2.z             = atomC.z - atomD.z;
            float3 cp0, cp1;
            float dihedralAngle, cosPhi;
      //      printf("%4d - 0 : %9.4f %9.4f %9.4f\n", pos1, A->v0.x, A->v0.y, A->v0.z);
      //      printf("%4d - 1 : %9.4f %9.4f %9.4f\n", pos1, A->v1.x, A->v1.y, A->v1.z);
      //      printf("%4d - 2 : %9.4f %9.4f %9.4f\n", pos1, A->v2.x, A->v2.y, A->v2.z);
            GETDIHEDRALANGLECOSINEBETWEENTHREEVECTORS(A->v0, A->v1, A->v2, A->v0, cp0, cp1, dihedralAngle, cosPhi);
            if (dihedralAngle < 0.0f )
            {
                dihedralAngle += LOCAL_HACK_PI;
            }
            else
            {
                dihedralAngle -= LOCAL_HACK_PI;
            }
            cosPhi                  = -cosPhi;
         //   printf("%4d: %9.4f %9.4f\n", pos1, dihedralAngle, cosPhi);
            float4 dihedral1        = cSim.pRbDihedralParameter1[pos1];
            float2 dihedral2        = cSim.pRbDihedralParameter2[pos1];
            float cosFactor         = cosPhi;
            float dEdAngle          = -dihedral1.y;

/* E */     float rb_energy         = dihedral1.x;
            rb_energy              += dihedral1.y * cosFactor;
        //    printf("%4d - 1: %9.4f %9.4f\n", pos1, dEdAngle, 1.0f);
            dEdAngle               -= 2.0f * dihedral1.z * cosFactor;
       //     printf("%4d - 2: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 3.0f * dihedral1.w * cosFactor;
            rb_energy              += dihedral1.z * cosFactor;
    //       printf("%4d - 3: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 4.0f * dihedral2.x * cosFactor;
            rb_energy              += dihedral1.w * cosFactor;
  //         printf("%4d - 4: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            cosFactor              *= cosPhi;
            dEdAngle               -= 5.0f * dihedral2.y * cosFactor;
            rb_energy              += dihedral2.x * cosFactor;
            rb_energy              += dihedral2.y * cosFactor * cosPhi;
/* E */     energy                 += rb_energy;
 //           printf("%4d - 5: %9.4f %9.4f\n", pos1, dEdAngle, cosFactor);
            dEdAngle               *= sinf(dihedralAngle);
//            printf("%4d - f: %9.4f\n", pos1, dEdAngle);

            float normCross1        = DOT3(cp0, cp0);
            float normBC            = sqrtf(DOT3(A->v1, A->v1));
            float4 ff;
            ff.x                    = (-dEdAngle * normBC) / normCross1;
            float normCross2        = DOT3(cp1, cp1);
            ff.w                    = (dEdAngle * normBC) / normCross2;
            float dp                = 1.0f / DOT3(A->v1, A->v1);
            ff.y                    = DOT3(A->v0, A->v1) * dp;
            ff.z                    = DOT3(A->v2, A->v1) * dp;
            int4  atom2             = cSim.pRbDihedralID2[pos1];
            float3 internalF0;
            float3 internalF3;
            float3 s;

//            printf("%4d: %9.4f %9.4f %9.4f %9.4f\n", pos1, ff.x, ff.y, ff.z, ff.w);
            unsigned int offset                 = atom1.x + atom2.x * cSim.stride;
            float4 force                        = cSim.pForce4[offset];
            internalF0.x                        = ff.x * cp0.x;
            force.x                            += internalF0.x;
            internalF0.y                        = ff.x * cp0.y;
            force.y                            += internalF0.y;
            internalF0.z                        = ff.x * cp0.z;
            force.z                            += internalF0.z;
            cSim.pForce4[offset]                = force;

 //           printf("%4d - 0: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.w + atom2.w * cSim.stride;
            force                               = cSim.pForce4[offset];
            internalF3.x                        = ff.w * cp1.x;
            force.x                            += internalF3.x;
            internalF3.y                        = ff.w * cp1.y;
            force.y                            += internalF3.y;
            internalF3.z                        = ff.w * cp1.z;
            force.z                            += internalF3.z;
            cSim.pForce4[offset]                = force;

   //         printf("%4d - 3: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            s.x                                 = ff.y * internalF0.x - ff.z * internalF3.x;
            s.y                                 = ff.y * internalF0.y - ff.z * internalF3.y;
            s.z                                 = ff.y * internalF0.z - ff.z * internalF3.z;
            offset                              = atom1.y + atom2.y * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF0.x + s.x;
            force.y                            += -internalF0.y + s.y;
            force.z                            += -internalF0.z + s.z;
            cSim.pForce4[offset]                = force;
     //       printf("%4d - 1: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
            offset                              = atom1.z + atom2.z * cSim.stride;
            force                               = cSim.pForce4[offset];
            force.x                            += -internalF3.x - s.x;
            force.y                            += -internalF3.y - s.y;
            force.z                            += -internalF3.z - s.z;
            cSim.pForce4[offset]                = force;
     //       printf("%4d - 2: %9.4f %9.4f %9.4f\n", pos1, cSim.pForce[offset], cSim.pForce[offset + cSim.stride], cSim.pForce[offset + cSim.stride2]);
        }         
        pos += blockDim.x * gridDim.x;
    }   

    while (pos < cSim.LJ14_offset)
    {
        unsigned int pos1       = pos - cSim.rb_dihedral_offset;
        if (pos1 < cSim.LJ14s)
        {
            int4 atom               = cSim.pLJ14ID[pos1];
            float4 LJ14             = cSim.pLJ14Parameter[pos1];
            float4 a1               = cSim.pPosq[atom.x];
            float4 a2               = cSim.pPosq[atom.y];
            float3 d;
            d.x                     = a1.x - a2.x;
            d.y                     = a1.y - a2.y;
            d.z                     = a1.z - a2.z;
            float r2                = DOT3(d, d);
            float inverseR          = 1.0f / sqrtf(r2);
            float sig2              = inverseR * LJ14.y;
            sig2                   *= sig2;
            float sig6              = sig2 * sig2 * sig2;
            float dEdR              = LJ14.x * (12.0f * sig6 - 6.0f) * sig6;
            /* E */
            energy                 += LJ14.x * (sig6 - 1.0f) * sig6;
            energy                 += LJ14.z * inverseR;

            dEdR                   += LJ14.z * inverseR;
            dEdR                   *= inverseR * inverseR;
            unsigned int offsetA    = atom.x + atom.z * cSim.stride;
            unsigned int offsetB    = atom.y + atom.w * cSim.stride;
            float4 forceA           = cSim.pForce4[offsetA];
            float4 forceB           = cSim.pForce4[offsetB];
            d.x                    *= dEdR;
            d.y                    *= dEdR;
            d.z                    *= dEdR;
            forceA.x               += d.x;
            forceA.y               += d.y;
            forceA.z               += d.z;
            forceB.x               -= d.x;
            forceB.y               -= d.y;
            forceB.z               -= d.z;
            cSim.pForce4[offsetA]   = forceA;
            cSim.pForce4[offsetB]   = forceB;
        }
        pos                    += blockDim.x * gridDim.x;
    }

    cSim.pEnergy[blockIdx.x * blockDim.x + threadIdx.x] += energy;
}


void kCalculateLocalForces(gpuContext gpu)
{
  //  printf("kCalculateLocalForces\n");
    kCalculateLocalForces_kernel<<<gpu->sim.blocks, gpu->sim.localForces_threads_per_block, gpu->sim.localForces_threads_per_block * sizeof(Vectors)>>>();
    LAUNCHERROR("kCalculateLocalForces");
}

