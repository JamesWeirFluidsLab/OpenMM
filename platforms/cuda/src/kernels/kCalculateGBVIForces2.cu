#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
using namespace std;

#include "gputypes.h"
#include "cudaKernels.h"

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float fx;
    float fy;
    float fz;
    float fb;
};


static __constant__ cudaGmxSimulation cSim;

void SetCalculateGBVIForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateGBVIForces2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

#include "kCalculateGBVIAux.h"

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateGBVIForces2.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateGBVIForces2.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateGBVIForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateGBVIForces2.h"

void kCalculateGBVIForces2(gpuContext gpu)
{

    switch (gpu->sim.nonbondedMethod)
    {
        case NO_CUTOFF:

            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVIN2ByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit );
            else
                kCalculateGBVIN2Forces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        sizeof(Atom)*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pWorkUnit );
            break;

        case CUTOFF:

            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVICutoffByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit );
            else
                kCalculateGBVICutoffForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            break;

        case PERIODIC:

            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVIPeriodicByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit );
            else
                kCalculateGBVIPeriodicForces2_kernel<<<gpu->sim.bornForce2_blocks, gpu->sim.bornForce2_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.bornForce2_threads_per_block>>>(gpu->sim.pInteractingWorkUnit );
            break;

    }
    LAUNCHERROR("kCalculateGBVIForces2");

    //kPrintGBVI( gpu, "kCalculateGBVIForces2", 0, stderr);
}
