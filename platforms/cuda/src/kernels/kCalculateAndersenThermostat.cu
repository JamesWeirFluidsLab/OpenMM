#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
//#include <fstream>
using namespace std;

#include "gputypes.h"

static __constant__ cudaGmxSimulation cSim;

void SetCalculateAndersenThermostatSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateAndersenThermostatSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

__global__ void kCalculateAndersenThermostat_kernel(int* atomGroups)
{
    unsigned int pos            = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rpos           = cSim.pRandomPosition[blockIdx.x];
    __syncthreads();

    float collisionProbability = 1.0f-exp(-cSim.collisionFrequency*cSim.pStepSize[0].y);
    float randomRange = erf(collisionProbability/sqrtf(2.0f));
    while (pos < cSim.atoms)
    {
        float4 velocity         = cSim.pVelm4[pos];
        float4 selectRand       = cSim.pRandom4[rpos + atomGroups[pos]];
        float4 velRand          = cSim.pRandom4[rpos + pos];
        float scale = (selectRand.w > -randomRange && selectRand.w < randomRange ? 0.0f : 1.0f);
        float add = (1.0f-scale)*sqrtf(cSim.kT*velocity.w);
        velocity.x = scale*velocity.x + add*velRand.x;
        velocity.y = scale*velocity.y + add*velRand.y;
        velocity.z = scale*velocity.z + add*velRand.z;
        cSim.pVelm4[pos]        = velocity;

        pos                    += blockDim.x * gridDim.x;
    }

    // Update random position pointer
    if (threadIdx.x == 0)
    {
        rpos                   += cSim.paddedNumberOfAtoms;
        if (rpos > cSim.randoms)
            rpos               -= cSim.randoms;
        cSim.pRandomPosition[blockIdx.x] = rpos;
    }
}

extern void kGenerateRandoms(gpuContext gpu);
void kCalculateAndersenThermostat(gpuContext gpu, CUDAStream<int>& atomGroups)
{
//    printf("kCalculateAndersenThermostat\n");
    kCalculateAndersenThermostat_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>(atomGroups._pDevData);
    LAUNCHERROR("kCalculateAndersenThermostat");
    
    // Update randoms if necessary
    gpu->iterations++;
    if (gpu->iterations == gpu->sim.randomIterations)
    {
        kGenerateRandoms(gpu);
        gpu->iterations = 0;
    }
}

