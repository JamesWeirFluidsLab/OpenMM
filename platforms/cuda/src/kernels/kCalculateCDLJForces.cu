#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"
#include "cudatypes.h"

#define UNROLLXX 0
#define UNROLLXY 0

struct Atom {
    float x;
    float y;
    float z;
    float q;
    float sig;
    float eps;
    float fx;
    float fy;
    float fz;
};

static __constant__ cudaGmxSimulation cSim;

void SetCalculateCDLJForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateCDLJForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

texture<float, 1, hipReadModeElementType> tabulatedErfcRef;

static __device__ float fastErfc(float r)
{
    float normalized = cSim.tabulatedErfcScale*r;
    int index = (int) normalized;
    float fract2 = normalized-index;
    float fract1 = 1.0f-fract2;
    return fract1*tex1Dfetch(tabulatedErfcRef, index) + fract2*tex1Dfetch(tabulatedErfcRef, index+1);
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateCDLJForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateCDLJForces.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateCDLJForces.h"
#include "kFindInteractingBlocks.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateCDLJForces.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateCDLJForces.h"
#include "kFindInteractingBlocks.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateCDLJForces.h"

// Include versions of the kernels for Ewald

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define USE_EWALD
#define METHOD_NAME(a, b) a##Ewald##b
#include "kCalculateCDLJForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##EwaldByWarp##b
#include "kCalculateCDLJForces.h"

// Reciprocal Space Ewald summation is in a separate kernel
#include "kCalculateCDLJEwaldFastReciprocal.h"

void kCalculatePME(gpuContext gpu);

void kCalculateCDLJForces(gpuContext gpu)
{
//    printf("kCalculateCDLJCutoffForces\n");
    switch (gpu->sim.nonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJN2ByWarpForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(Atom)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pWorkUnit);
            else
                kCalculateCDLJN2Forces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(Atom)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pWorkUnit);
            LAUNCHERROR("kCalculateCDLJN2Forces");
            break;
        case CUTOFF:
            kFindBlockBoundsCutoff_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsCutoff");
            kFindBlocksWithInteractionsCutoff_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsCutoff");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksCutoff_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJCutoffByWarpForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJCutoffForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJCutoffForces");
            break;
        case PERIODIC:
            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJPeriodicByWarpForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJPeriodicForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJPeriodicForces");
            break;
        case EWALD:
        case PARTICLE_MESH_EWALD:
            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kFindInteractionsWithinBlocksPeriodic");
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            hipBindTexture(NULL, &tabulatedErfcRef, gpu->psTabulatedErfc->_pDevData, &channelDesc, gpu->psTabulatedErfc->_length*sizeof(float));
            if (gpu->bOutputBufferPerWarp)
                kCalculateCDLJEwaldByWarpForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCDLJEwaldForces_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        (sizeof(Atom)+sizeof(float3))*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCDLJEwaldForces");
            if (gpu->sim.nonbondedMethod == EWALD)
            {
                kCalculateEwaldFastCosSinSums_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block>>>();
                LAUNCHERROR("kCalculateEwaldFastCosSinSums");
                kCalculateEwaldFastForces_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
                LAUNCHERROR("kCalculateEwaldFastForces");
            }
            else
                kCalculatePME(gpu);
    }
}
