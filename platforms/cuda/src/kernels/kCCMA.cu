#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <vector>
#include "gputypes.h"

using namespace std;


static __constant__ cudaGmxSimulation cSim;

void SetCCMASim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCCMASim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

__global__ void
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(1024, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(512, 1)
#else
__launch_bounds__(256, 1)
#endif
kComputeCCMAConstraintDirections()
{
    // Calculate the direction of each constraint.

    for (unsigned int index = threadIdx.x+blockIdx.x*blockDim.x; index < cSim.ccmaConstraints; index += blockDim.x*gridDim.x)
    {
        int2 atoms = cSim.pCcmaAtoms[index];
        float4 dir = cSim.pCcmaDistance[index];
        float4 oldPos1 = cSim.pOldPosq[atoms.x];
        float4 oldPos2 = cSim.pOldPosq[atoms.y];
        dir.x = oldPos1.x-oldPos2.x;
        dir.y = oldPos1.y-oldPos2.y;
        dir.z = oldPos1.z-oldPos2.z;
        cSim.pCcmaDistance[index] = dir;
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(1024, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(512, 1)
#else
__launch_bounds__(256, 1)
#endif
kComputeCCMAConstraintForces(float4* atomPositions, bool addOldPosition)
{
    __shared__ int converged;
    float lowerTol = 1.0f-2.0f*cSim.shakeTolerance+cSim.shakeTolerance*cSim.shakeTolerance;
    float upperTol = 1.0f+2.0f*cSim.shakeTolerance+cSim.shakeTolerance*cSim.shakeTolerance;
    if (threadIdx.x == 0)
        converged = 1;
    __syncthreads();

    // Calculate the constraint force for each constraint.

    for (unsigned int index = threadIdx.x+blockIdx.x*blockDim.x; index < cSim.ccmaConstraints; index += blockDim.x*gridDim.x)
    {
        int2 atoms = cSim.pCcmaAtoms[index];
        float4 delta1 = atomPositions[atoms.x];
        float4 delta2 = atomPositions[atoms.y];
        float4 dir = cSim.pCcmaDistance[index];
        float3 rp_ij = make_float3(delta1.x-delta2.x, delta1.y-delta2.y, delta1.z-delta2.z);
        if (addOldPosition)
        {
            rp_ij.x += dir.x;
            rp_ij.y += dir.y;
            rp_ij.z += dir.z;
        }
        float rp2 = rp_ij.x*rp_ij.x + rp_ij.y*rp_ij.y + rp_ij.z*rp_ij.z;
        float dist2 = dir.w*dir.w;
        float diff = dist2 - rp2;
        float rrpr  = rp_ij.x*dir.x + rp_ij.y*dir.y + rp_ij.z*dir.z;
        float d_ij2  = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z;
        float reducedMass = cSim.pCcmaReducedMass[index];
        cSim.pCcmaDelta1[index] = (rrpr > d_ij2*1e-6f ? reducedMass*diff/rrpr : 0.0f);

        // See whether it has converged.

        if (converged && (rp2 < lowerTol*dist2 || rp2 > upperTol*dist2))
        {
            converged = 0;
            *cSim.ccmaConvergedDeviceMarker = 0;
        }
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(1024, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(512, 1)
#else
__launch_bounds__(256, 1)
#endif
kMultiplyByCCMAConstraintMatrix()
{
    if (*cSim.ccmaConvergedDeviceMarker)
        return; // The constraint iteration has already converged

    // Multiply by the inverse constraint matrix.

    for (unsigned int index = threadIdx.x+blockIdx.x*blockDim.x; index < cSim.ccmaConstraints; index += blockDim.x*gridDim.x)
    {
        float sum = 0.0f;
        for (unsigned int i = 0; ; i++)
        {
            unsigned int element = index+i*cSim.ccmaConstraints;
            unsigned int column = cSim.pConstraintMatrixColumn[element];
            if (column >= cSim.ccmaConstraints)
                break;
            sum += cSim.pCcmaDelta1[column]*cSim.pConstraintMatrixValue[element];
        }
        cSim.pCcmaDelta2[index] = sum;
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(1024, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(512, 1)
#else
__launch_bounds__(256, 1)
#endif
kUpdateCCMAAtomPositions(float4* atomPositions, int iteration)
{
    if (*cSim.ccmaConvergedDeviceMarker)
        return; // The constraint iteration has already converged.
    float damping = (iteration < 2 ? 0.5f : 1.0f);
    for (unsigned int index = threadIdx.x+blockIdx.x*blockDim.x; index < cSim.atoms; index += blockDim.x*gridDim.x)
    {
        float4 atomPos = atomPositions[index];
        float invMass = cSim.pVelm4[index].w;
        int num = cSim.pCcmaNumAtomConstraints[index];
        for (int i = 0; i < num; i++)
        {
            int constraint = cSim.pCcmaAtomConstraints[index+i*cSim.atoms];
            bool forward = (constraint > 0);
            constraint = (forward ? constraint-1 : -constraint-1);
            float constraintForce = damping*invMass*cSim.pCcmaDelta2[constraint];
            constraintForce = (forward ? constraintForce : -constraintForce);
            float4 dir = cSim.pCcmaDistance[constraint];
            atomPos.x += constraintForce*dir.x;
            atomPos.y += constraintForce*dir.y;
            atomPos.z += constraintForce*dir.z;
        }
        atomPositions[index] = atomPos;
    }
}

void kApplyCCMA(gpuContext gpu, float4* posq, bool addOldPosition)
{
    kComputeCCMAConstraintDirections<<<gpu->sim.blocks, gpu->sim.ccma_threads_per_block>>>();
    LAUNCHERROR("kComputeCCMAConstraintDirections");
    const int checkInterval = 3;
    for (int i = 0; i < 150; i++) {
        if ((i+1)%checkInterval == 0)
            *gpu->ccmaConvergedHostMarker = 1;
        kComputeCCMAConstraintForces<<<gpu->sim.blocks, gpu->sim.ccma_threads_per_block, gpu->sim.ccma_threads_per_block*sizeof(int)>>>(posq, addOldPosition);
        hipEventRecord(gpu->ccmaEvent, 0);
        kMultiplyByCCMAConstraintMatrix<<<gpu->sim.blocks, gpu->sim.ccma_threads_per_block, gpu->sim.ccma_threads_per_block*sizeof(int)>>>();
        kUpdateCCMAAtomPositions<<<gpu->sim.blocks, gpu->sim.ccma_threads_per_block>>>(posq, 3*i+2);
        hipEventSynchronize(gpu->ccmaEvent);
        if ((i+1)%checkInterval == 0 && *gpu->ccmaConvergedHostMarker)
            break;
    }
}

void kApplyCCMA(gpuContext gpu)
{
//    printf("kApplyCCMA\n");
    if (gpu->sim.ccmaConstraints > 0)
        kApplyCCMA(gpu, gpu->sim.pPosqP, true);
}
