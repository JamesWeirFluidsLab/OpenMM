#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
using namespace std;

#include "gputypes.h"

enum {VelScale, ForceScale, NoiseScale, MaxParams};

static __constant__ cudaGmxSimulation cSim;

void SetLangevinUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetLangevinUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));     
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

// Include versions of the kernels with and with center of mass motion removal.

#include "kLangevinUpdate.h"
#define REMOVE_CM
#include "kLangevinUpdate.h"

void kLangevinUpdatePart1(gpuContext gpu)
{
//    printf("kLangevinUpdatePart1\n");
    if (gpu->bRemoveCM)
    {
        kLangevinUpdatePart1CM_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block, gpu->sim.update_threads_per_block * sizeof(float3)>>>();
        LAUNCHERROR("kLangevinUpdatePart1CM");
        gpu->bRemoveCM = false;
    }
    else
    {    
        kLangevinUpdatePart1_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
        LAUNCHERROR("kLangevinUpdatePart1");
    }
}

extern void kGenerateRandoms(gpuContext gpu);
void kLangevinUpdatePart2(gpuContext gpu)
{
//    printf("kLangevinUpdatePart2\n");
    if (gpu->bCalculateCM)
    {
        kLangevinUpdatePart2CM_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block, gpu->sim.update_threads_per_block * sizeof(float3)>>>();
        LAUNCHERROR("kLangevinUpdatePart2CM");
        gpu->bCalculateCM = false;
        gpu->bRemoveCM = true;
    }
    else
    {
        kLangevinUpdatePart2_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
        LAUNCHERROR("kLangevinUpdatePart2");
    }
    
    // Update randoms if necessary
    gpu->iterations++;
    if (gpu->iterations == gpu->sim.randomIterations)
    {
        kGenerateRandoms(gpu);
        gpu->iterations = 0;
    }
}


__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_UPDATE_THREADS_PER_BLOCK, 1)
#endif
void kSelectLangevinStepSize_kernel(float maxStepSize)
{
    // Calculate the error.

    extern __shared__ float error[];
    __shared__ float params[MaxParams];
    error[threadIdx.x] = 0.0f;
    unsigned int pos = threadIdx.x;
    while (pos < cSim.atoms)
    {
        float4 force  = cSim.pForce4[pos];
        float invMass = cSim.pVelm4[pos].w;
        error[threadIdx.x] += (force.x*force.x + force.y*force.y + force.z*force.z)*invMass;
        pos += blockDim.x * gridDim.x;
    }
    __syncthreads();

    // Sum the errors from all threads.

    for (int offset = 1; offset < blockDim.x; offset *= 2)
    {
        if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0)
            error[threadIdx.x] += error[threadIdx.x+offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        // Select the new step size.
        
        float totalError = sqrt(error[0]/(cSim.atoms*3));
        float newStepSize = sqrt(cSim.errorTol/totalError);
        float oldStepSize = cSim.pStepSize[0].y;
        if (oldStepSize > 0.0f)
            newStepSize = min(newStepSize, oldStepSize*2.0f); // For safety, limit how quickly dt can increase.
        if (newStepSize > oldStepSize && newStepSize < 1.1f*oldStepSize)
            newStepSize = oldStepSize; // Keeping dt constant between steps improves the behavior of the integrator.
        if (newStepSize > maxStepSize)
            newStepSize = maxStepSize;
        cSim.pStepSize[0].y = newStepSize;

        // Recalculate the integration parameters.

        float vscale = exp(-newStepSize/cSim.tau);
        float fscale = (1-vscale)*cSim.tau;
        float noisescale = sqrt(2*cSim.kT/cSim.tau)*sqrt(0.5f*(1-vscale*vscale)*cSim.tau);
        params[VelScale] = vscale;
        params[ForceScale] = fscale;
        params[NoiseScale] = noisescale;
    }
    __syncthreads();
    if (threadIdx.x < MaxParams)
        cSim.pLangevinParameters[threadIdx.x] = params[threadIdx.x];
}

void kSelectLangevinStepSize(gpuContext gpu, float maxTimeStep)
{
//    printf("kSelectLangevinStepSize\n");
    kSelectLangevinStepSize_kernel<<<1, gpu->sim.update_threads_per_block, sizeof(float)*gpu->sim.update_threads_per_block>>>(maxTimeStep);
    LAUNCHERROR("kSelectLangevinStepSize");
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_UPDATE_THREADS_PER_BLOCK, 1)
#endif
void kSetVelocitiesFromPositions_kernel()
{
    float2 stepSize = cSim.pStepSize[0];
    double oneOverDt = 2.0/(stepSize.x+stepSize.y);
    unsigned int pos = threadIdx.x;
    while (pos < cSim.atoms)
    {
        float4 posq = cSim.pPosq[pos];
        float4 posqP = cSim.pPosqP[pos];
        float4 velm = cSim.pVelm4[pos];
        velm.x = (float) (oneOverDt*posqP.x);
        velm.y = (float) (oneOverDt*posqP.y);
        velm.z = (float) (oneOverDt*posqP.z);
        cSim.pVelm4[pos] = velm;
        posq.x += posqP.x;
        posq.y += posqP.y;
        posq.z += posqP.z;
        cSim.pPosq[pos] = posq;
        pos += blockDim.x * gridDim.x;
    }
}

void kSetVelocitiesFromPositions(gpuContext gpu)
{
//    printf("kSetVelocitiesFromPositions\n");
    kSetVelocitiesFromPositions_kernel<<<1, gpu->sim.update_threads_per_block, sizeof(float)*gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kSetVelocitiesFromPositions");
}
