#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

#include "gputypes.h"
#include "cudatypes.h"

#define UNROLLXX 0
#define UNROLLXY 0

struct Atom {
    float x;
    float y;
    float z;
    float4 params;
    float fx;
    float fy;
    float fz;
};

static __constant__ cudaGmxSimulation cSim;
static __constant__ Expression<256> forceExp;
static __constant__ Expression<256> energyExp;

#include "kEvaluateExpression.h"

void SetCalculateCustomNonbondedForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetCalculateCustomNonbondedForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyFromSymbol: SetSim copy from cSim failed");
}

void SetCustomNonbondedForceExpression(const Expression<256>& expression)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(forceExp), &expression, sizeof(forceExp));
    RTERROR(status, "SetCustomNonbondedForceExpression: hipMemcpyToSymbol failed");
}

void SetCustomNonbondedEnergyExpression(const Expression<256>& expression)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(energyExp), &expression, sizeof(energyExp));
    RTERROR(status, "SetCustomNonbondedEnergyExpression: hipMemcpyToSymbol failed");
}

void SetCustomNonbondedGlobalParams(const vector<float>& paramValues)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(globalParams), &paramValues[0], paramValues.size()*sizeof(float));
    RTERROR(status, "SetCustomNonbondedGlobalParams: hipMemcpyToSymbol failed");
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateCustomNonbondedForces.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateCustomNonbondedForces.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateCustomNonbondedForces.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateCustomNonbondedForces.h"

__global__ void kFindBlockBoundsCutoff_kernel();
__global__ void kFindBlocksWithInteractionsCutoff_kernel();
__global__ void kFindInteractionsWithinBlocksCutoff_kernel(unsigned int* workUnit);
__global__ void kFindBlockBoundsPeriodic_kernel();
__global__ void kFindBlocksWithInteractionsPeriodic_kernel();
__global__ void kFindInteractionsWithinBlocksPeriodic_kernel(unsigned int* workUnit);

void kCalculateCustomNonbondedForces(gpuContext gpu, bool neighborListValid)
{
//    printf("kCalculateCustomNonbondedCutoffForces\n");
    if (gpu->tabulatedFunctionsChanged)
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        if (gpu->tabulatedFunctions[0].coefficients != NULL)
            hipBindTexture(NULL, &texRef0, gpu->tabulatedFunctions[0].coefficients->_pDevData, &channelDesc, gpu->tabulatedFunctions[0].coefficients->_length*sizeof(float4));
        if (gpu->tabulatedFunctions[1].coefficients != NULL)
            hipBindTexture(NULL, &texRef1, gpu->tabulatedFunctions[1].coefficients->_pDevData, &channelDesc, gpu->tabulatedFunctions[1].coefficients->_length*sizeof(float4));
        if (gpu->tabulatedFunctions[2].coefficients != NULL)
            hipBindTexture(NULL, &texRef2, gpu->tabulatedFunctions[2].coefficients->_pDevData, &channelDesc, gpu->tabulatedFunctions[2].coefficients->_length*sizeof(float4));
        if (gpu->tabulatedFunctions[3].coefficients != NULL)
            hipBindTexture(NULL, &texRef3, gpu->tabulatedFunctions[3].coefficients->_pDevData, &channelDesc, gpu->tabulatedFunctions[3].coefficients->_length*sizeof(float4));
        gpu->tabulatedFunctionsChanged = false;
    }
    int sharedPerThread = sizeof(Atom)+gpu->sim.customExpressionStackSize*sizeof(float)+9*sizeof(float);
    if (gpu->sim.customNonbondedMethod != NO_CUTOFF)
        sharedPerThread += sizeof(float3);
    int threads = gpu->sim.nonbond_threads_per_block;
    int maxThreads = (gpu->sharedMemoryPerBlock-16)/sharedPerThread;
    if (threads > maxThreads)
        threads = (maxThreads/32)*32;
    switch (gpu->sim.customNonbondedMethod)
    {
        case NO_CUTOFF:
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedN2ByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pWorkUnit);
            else
                kCalculateCustomNonbondedN2Forces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedN2Forces");
            break;
        case CUTOFF:
            if (!neighborListValid)
            {
                kFindBlockBoundsCutoff_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
                LAUNCHERROR("kFindBlockBoundsCutoff");
                kFindBlocksWithInteractionsCutoff_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
                LAUNCHERROR("kFindBlocksWithInteractionsCutoff");
                compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
                kFindInteractionsWithinBlocksCutoff_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedCutoffByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCustomNonbondedCutoffForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedCutoffForces");
            break;
        case PERIODIC:
            if (!neighborListValid)
            {
                kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
                LAUNCHERROR("kFindBlockBoundsPeriodic");
                kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
                LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
                compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
                kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                        sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
            }
            if (gpu->bOutputBufferPerWarp)
                kCalculateCustomNonbondedPeriodicByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateCustomNonbondedPeriodicForces_kernel<<<gpu->sim.nonbond_blocks, threads, sharedPerThread*threads>>>(gpu->sim.pInteractingWorkUnit);
            LAUNCHERROR("kCalculateCustomNonbondedPeriodicForces");
            break;
    }
}
