#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * Permission is hereby granted, free of charge, to any person obtaining a    *
 * copy of this software and associated documentation files (the "Software"), *
 * to deal in the Software without restriction, including without limitation  *
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,   *
 * and/or sell copies of the Software, and to permit persons to whom the      *
 * Software is furnished to do so, subject to the following conditions:       *
 *                                                                            *
 * The above copyright notice and this permission notice shall be included in *
 * all copies or substantial portions of the Software.                        *
 *                                                                            *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR *
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,   *
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL    *
 * THE AUTHORS, CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,    *
 * DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR      *
 * OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE  *
 * USE OR OTHER DEALINGS IN THE SOFTWARE.                                     *
 * -------------------------------------------------------------------------- */

#include "GpuFreeEnergyCudaKernels.h"
#include "freeEnergyGpuTypes.h"

#include "openmm/OpenMMException.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>

#define PARAMETER_PRINT 0
#define MAX_PARAMETER_PRINT 10

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaFreeEnergyGmxSimulation gbviSimDev;

void SetCalculateGBVISoftcoreBornSumGpuSim( freeEnergyGpuContext freeEnergyGpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL( cSim), &freeEnergyGpu->gpuContext->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateGBVISoftcoreBornSumGpuSim copy to cSim failed");

    status = hipMemcpyToSymbol(HIP_SYMBOL( gbviSimDev), &freeEnergyGpu->freeEnergySim, sizeof(cudaFreeEnergyGmxSimulation));    
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateGBVISoftcoreBornSumGpuSim copy to feSim failed");
}

// create, initialize and enter BornRadiusScaleFactors values (used to scale contribution of atoms to Born sum of other atoms)

extern "C"
void gpuSetGBVISoftcoreParameters( freeEnergyGpuContext freeEnergyGpu, float innerDielectric, float solventDielectric, const std::vector<int>& atom,
                                   const std::vector<float>& radius, const std::vector<float>& gamma,
                                   const std::vector<float>& scaledRadii, const std::vector<float>& bornRadiusScaleFactors,
                                   const std::vector<float>& quinticSplineParameters ){

    unsigned int numberOfParticles                    = radius.size();

    gpuContext gpu                                    = freeEnergyGpu->gpuContext;

    static const float electricConstant               = -166.02691f;
    double tau                                        = ((1.0f/innerDielectric)-(1.0f/solventDielectric)); 
    freeEnergyGpu->psSwitchDerivative                 = new CUDAStream<float>( gpu->sim.paddedNumberOfAtoms, 1, "SwitchDerivative");
    freeEnergyGpu->freeEnergySim.pSwitchDerivative    = freeEnergyGpu->psSwitchDerivative->_pDevData;

    // create gpuGBVISoftcore, load parameters, and track minimum softcore value
    // gpuGBVISoftcore is not really being used (it was in the initial implementation) -- 
    // will be removed in future once confirmed not needed


    // check if quintic scaling to be applied

    if( quinticSplineParameters.size() == 2 ){
       freeEnergyGpu->freeEnergySim.bornRadiiScalingMethod        = 1;
       freeEnergyGpu->freeEnergySim.quinticLowerLimitFactor       = quinticSplineParameters[0];
       freeEnergyGpu->freeEnergySim.quinticUpperLimit             = quinticSplineParameters[1];
    } else {
       freeEnergyGpu->freeEnergySim.bornRadiiScalingMethod        = 0;
       freeEnergyGpu->freeEnergySim.quinticLowerLimitFactor       = 0.8f;
       freeEnergyGpu->freeEnergySim.quinticUpperLimit             = 5.0f;
    }

    for( unsigned int ii = 0; ii < bornRadiusScaleFactors.size(); ii++ ){
            (*gpu->psGBVIData)[ii].x                 = radius[ii];
            (*gpu->psGBVIData)[ii].y                 = scaledRadii[ii];
            (*gpu->psGBVIData)[ii].z                 = tau*gamma[ii];
            (*gpu->psGBVIData)[ii].w                 = bornRadiusScaleFactors[ii];
            (*gpu->psBornRadii)[ii]                  = 0.0f;
            (*freeEnergyGpu->psSwitchDerivative)[ii] = 0.0f;
    }

    // Dummy out extra atom data

    for( unsigned int ii = bornRadiusScaleFactors.size(); ii < gpu->sim.paddedNumberOfAtoms; ii++ ){
        (*gpu->psGBVIData)[ii].x                 = 0.01f;
        (*gpu->psGBVIData)[ii].y                 = 0.01f;
        (*gpu->psGBVIData)[ii].z                 = 0.0f;
        (*gpu->psGBVIData)[ii].w                 = 0.0f;
        (*gpu->psBornRadii)[ii]                  = 0.0f;
        (*freeEnergyGpu->psSwitchDerivative)[ii] = 0.0f;
    }

    gpu->sim.preFactor               = 2.0f*electricConstant*((1.0f/innerDielectric)-(1.0f/solventDielectric))*gpu->sim.forceConversionFactor;

    // diagnostics

    if( freeEnergyGpu->log ){
        (void) fprintf( freeEnergyGpu->log,"GBVISoftcore: part.=%u padded=%u sclMeth=%d\n",
                        static_cast<unsigned int>(bornRadiusScaleFactors.size()), static_cast<unsigned int>(gpu->sim.paddedNumberOfAtoms),
                        freeEnergyGpu->freeEnergySim.bornRadiiScalingMethod );
        if( quinticSplineParameters.size() == 2 ){
            (void) fprintf( freeEnergyGpu->log,"QuinticScaling: LwFct=%8.3f UpLmt=[%12.5e (nm) %12.5e]\n",
                        freeEnergyGpu->freeEnergySim.quinticLowerLimitFactor,
                        powf( freeEnergyGpu->freeEnergySim.quinticUpperLimit, -0.3333333f ), freeEnergyGpu->freeEnergySim.quinticUpperLimit );
        }
        (void) fprintf( freeEnergyGpu->log, "gpuSetGBVISoftcoreParameters: preFactor=%14.6e elecCnstnt=%.4f frcCnvrsnFctr=%.4f tau=%.4f.\n",
                        gpu->sim.preFactor, 2.0f*electricConstant, gpu->sim.forceConversionFactor, ((1.0f/innerDielectric)-(1.0f/solventDielectric)) );
#ifdef PARAMETER_PRINT
        int maxPrint = MAX_PARAMETER_PRINT;
        (void) fprintf( freeEnergyGpu->log, "               radius  scaled radius      tau*gamma         lambda\n" );
        for( unsigned int ii = 0; ii < bornRadiusScaleFactors.size(); ii++ ){
    
            (void) fprintf( freeEnergyGpu->log,"%6u %14.7e %14.7e %14.7e %14.7e\n",
                            ii, (*gpu->psGBVIData)[ii].x, (*gpu->psGBVIData)[ii].y, (*gpu->psGBVIData)[ii].z, (*gpu->psGBVIData)[ii].w ); 
            if( ii == maxPrint ){
                ii = bornRadiusScaleFactors.size() - maxPrint;
                if( ii < maxPrint )ii = maxPrint;
            }
        }
        unsigned int offset = gpu->sim.paddedNumberOfAtoms - MAX_PARAMETER_PRINT;
        if( offset > 0 && gpu->sim.paddedNumberOfAtoms > bornRadiusScaleFactors.size()  ){
            for( unsigned int ii = offset; ii < gpu->sim.paddedNumberOfAtoms; ii++ ){
                (void) fprintf( freeEnergyGpu->log,"%6u %14.7e %14.7e %14.7e %14.7e\n",
                                ii, (*gpu->psGBVIData)[ii].x, (*gpu->psGBVIData)[ii].y, (*gpu->psGBVIData)[ii].z, (*gpu->psGBVIData)[ii].w ); 
            }
        }
#endif
    }

    gpu->psGBVIData->Upload();
    gpu->psBornRadii->Upload();
    freeEnergyGpu->psSwitchDerivative->Upload();

    return;
}

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float sum;
    float gamma;
    float bornRadiusScaleFactor;
};

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kClearGBVISoftcoreBornSum_kernel()
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    while (pos < cSim.stride * cSim.nonbondOutputBuffers)
    {   
        ((float*)cSim.pBornSum)[pos]  = 0.0f;
        pos                          += gridDim.x * blockDim.x;
    }   
}

void kClearGBVISoftcoreBornSum(gpuContext gpu) {
    kClearGBVISoftcoreBornSum_kernel<<<gpu->sim.blocks, gpu->sim.threads_per_block>>>();
}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kReduceGBVISoftcoreBornForces_kernel()
{
    unsigned int pos = (blockIdx.x * blockDim.x + threadIdx.x);
    float energy     = 0.0f;
    while (pos < cSim.atoms)
    {
        float bornRadius  = cSim.pBornRadii[pos];
        float4 gbviData   = cSim.pGBVIData[pos];
        float totalForce  = 0.0f;
        float* pFt        = cSim.pBornForce + pos;

        int i = cSim.nonbondOutputBuffers;
        while (i >= 4)
        {
            float f1    = *pFt;
            pFt        += cSim.stride;
            float f2    = *pFt;
            pFt        += cSim.stride;
            float f3    = *pFt;
            pFt        += cSim.stride;
            float f4    = *pFt;
            pFt        += cSim.stride;
            totalForce += f1 + f2 + f3 + f4;
            i -= 4;
        }
        if (i >= 2)
        {
            float f1    = *pFt;
            pFt        += cSim.stride;
            float f2    = *pFt;
            pFt        += cSim.stride;
            totalForce += f1 + f2;
            i -= 2;
        }
        if (i > 0)
        {
            totalForce += *pFt;
        }

        float ratio         = (gbviData.x/bornRadius);
        float ratio3        = ratio*ratio*ratio;

        energy             -= gbviData.z*ratio3;                   //  gbviData.z = gamma*tau

        totalForce         += (3.0f*gbviData.z*ratio3)/bornRadius; // 'cavity' term
        float br2           = bornRadius*bornRadius;
        totalForce         *= (1.0f/3.0f)*br2*br2;

        pFt                 = cSim.pBornForce + pos;
        *pFt                = totalForce;
        pos                += gridDim.x * blockDim.x;
    }
    cSim.pEnergy[blockIdx.x * blockDim.x + threadIdx.x] += energy;
}

void kReduceGBVISoftcoreBornForces( freeEnergyGpuContext freeEnergyGpu )
{
    gpuContext gpu = freeEnergyGpu->gpuContext;
    kReduceGBVISoftcoreBornForces_kernel<<<gpu->sim.blocks, gpu->sim.bf_reduce_threads_per_block>>>();
    LAUNCHERROR("kReduceGBVISoftcoreBornForces");

}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kReduceGBVISoftcoreBornSum_kernel()
{
    unsigned int pos = (blockIdx.x * blockDim.x + threadIdx.x);
    
    while (pos < cSim.atoms)
    {
        float sum    = 0.0f;
        float* pSt   = cSim.pBornSum + pos;
        float4 atom  = cSim.pGBVIData[pos];
        
        // Get summed Born data
        for (int i = 0; i < cSim.nonbondOutputBuffers; i++)
        {
            sum += *pSt;
            pSt += cSim.stride;
        }
        
        // Now calculate Born radius

        float Rinv           = 1.0f/atom.x;
        sum                  = Rinv*Rinv*Rinv - sum; 
        cSim.pBornRadii[pos] = pow( sum, (-1.0f/3.0f) ); 
        pos                 += gridDim.x * blockDim.x;
    }   
}

void kReduceGBVISoftcoreBornSum( freeEnergyGpuContext freeEnergyGpu )
{
    gpuContext gpu = freeEnergyGpu->gpuContext;
    kReduceGBVISoftcoreBornSum_kernel<<<gpu->sim.blocks, gpu->sim.threads_per_block>>>();
    LAUNCHERROR("kReduceGBVISoftcoreBornSum");
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateGBVISoftcoreBornSum.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateGBVISoftcoreBornSum.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateGBVISoftcoreBornSum.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateGBVISoftcoreBornSum.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateGBVISoftcoreBornSum.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateGBVISoftcoreBornSum.h"

 __device__ void quinticSpline( float  x, float rl, float ru, float* outValue, float* outDerivative )
{
   float numerator    = x  - rl;
   float denominator  = ru - rl;
   float ratio        = numerator/denominator;
   float ratio2       = ratio*ratio;
   float ratio3       = ratio2*ratio;

   *outValue          =  1.0f + ratio3*(-10.f + 3.0f*ratio*(5.0f - 2.0f*ratio));
   *outDerivative     = -30.0f*ratio2*( 1.0f + ratio*(ratio - 2.0f))/denominator;
}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kReduceGBVIBornSumQuinticScaling_kernel()
{
    unsigned int pos = (blockIdx.x * blockDim.x + threadIdx.x);
    
    while (pos < cSim.atoms)
    {
        float sum   = 0.0f;
        float* pSt  = cSim.pBornSum + pos;
        float4 atom = cSim.pGBVIData[pos];
        
        // Get summed Born data
        for (int i = 0; i < cSim.nonbondOutputBuffers; i++)
        {
            sum += *pSt;
            pSt += cSim.stride;
        }
        
        // Now calculate Born radius

        float Rinv           = 1.0f/atom.x;
        float r3             = Rinv*Rinv*Rinv;
        float splineL        = gbviSimDev.quinticLowerLimitFactor*r3;
        float switchDeriviative;
        if( sum > splineL ){
            if( sum < r3 ){
                float splineValue, splineDerivative;
                quinticSpline( sum, splineL, r3, &splineValue, &splineDerivative ); 
                switchDeriviative  = splineValue - (r3 - sum)*splineDerivative;
                sum                = (r3 - sum)*splineValue + gbviSimDev.quinticUpperLimit;
            } else {
                sum                = gbviSimDev.quinticUpperLimit;
                switchDeriviative  = 0.0f;
            }
        } else {
            sum               = r3 - sum;
            switchDeriviative = 1.0f;
        }

        cSim.pBornRadii[pos]               = pow( sum, (-1.0f/3.0f) ); 
        gbviSimDev.pSwitchDerivative[pos]  = switchDeriviative;
        pos                               += gridDim.x * blockDim.x;
    }   
}

void kReduceGBVIBornSumQuinticScaling( freeEnergyGpuContext freeEnergyGpu )
{
    gpuContext gpu = freeEnergyGpu->gpuContext;
    kReduceGBVIBornSumQuinticScaling_kernel<<<gpu->sim.blocks, gpu->sim.threads_per_block>>>();
    LAUNCHERROR("kReduceGBVIBornSumQuinticScaling_kernel");
}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kReduceGBVIBornForcesQuinticScaling_kernel()
{
    unsigned int pos = (blockIdx.x * blockDim.x + threadIdx.x);
    float energy     = 0.0f;
    while (pos < cSim.atoms)
    {
        float bornRadius    = cSim.pBornRadii[pos];
        float4 gbviData     = cSim.pGBVIData[pos];
        float  switchDeriv  = gbviSimDev.pSwitchDerivative[pos];
        float totalForce    = 0.0f;
        float* pFt          = cSim.pBornForce + pos;

        int i = cSim.nonbondOutputBuffers;
        while (i >= 4)
        {
            float f1    = *pFt;
            pFt        += cSim.stride;
            float f2    = *pFt;
            pFt        += cSim.stride;
            float f3    = *pFt;
            pFt        += cSim.stride;
            float f4    = *pFt;
            pFt        += cSim.stride;
            totalForce += f1 + f2 + f3 + f4;
            i -= 4;
        }
        if (i >= 2)
        {
            float f1    = *pFt;
            pFt        += cSim.stride;
            float f2    = *pFt;
            pFt        += cSim.stride;
            totalForce += f1 + f2;
            i -= 2;
        }
        if (i > 0)
        {
            totalForce += *pFt;
        }

        float ratio         = (gbviData.x/bornRadius);
        float ratio3        = ratio*ratio*ratio;
        energy             -= gbviData.z*ratio3;
        totalForce         += (3.0f*gbviData.z*ratio3)/bornRadius; // 'cavity' term
        float br2           = bornRadius*bornRadius;
        totalForce         *= (1.0f/3.0f)*br2*br2*switchDeriv;

        pFt                 = cSim.pBornForce + pos;
        *pFt                = totalForce;
        pos                += gridDim.x * blockDim.x;
    }
    cSim.pEnergy[blockIdx.x * blockDim.x + threadIdx.x] += energy;
}

void kReduceGBVIBornForcesQuinticScaling( freeEnergyGpuContext freeEnergyGpu )
{
    gpuContext gpu = freeEnergyGpu->gpuContext;

    //(void) fprintf( stderr, "kReduceObcGbsaBornForces %6d blks=%u bsf_reduce_threads_per_block=%5u %5u   %5u %5u %5u\n",
    //                 gpu->natoms, gpu->sim.blocks, gpu->sim.bsf_reduce_threads_per_block, gpu->sim.bf_reduce_threads_per_block,
    //                 GF1XX_THREADS_PER_BLOCK, GT2XX_THREADS_PER_BLOCK, G8X_THREADS_PER_BLOCK); fflush( stderr );
    kReduceGBVIBornForcesQuinticScaling_kernel<<<gpu->sim.blocks, gpu->sim.bsf_reduce_threads_per_block>>>();
    LAUNCHERROR("kReduceGBVIBornForcesQuinticScaling");
}

void kPrintGBVISoftcore( freeEnergyGpuContext freeEnergyGpu, std::string callId, int call, FILE* log)
{
    gpuContext gpu = freeEnergyGpu->gpuContext;
    //int maxPrint   = gpu->natoms;

    gpu->psGBVIData->Download();
    gpu->psBornRadii->Download();
    gpu->psBornForce->Download();
    gpu->psPosq4->Download();

    CUDAStream<float>* switchDeriviative = freeEnergyGpu->psSwitchDerivative;
    CUDAStream<float4>* sigEps4          = freeEnergyGpu->psSigEps4;

    switchDeriviative->Download();
    sigEps4->Download();

    (void) fprintf( log, "kPrintGBVISoftcore Cuda Softcore bR bF swd   prm    sigeps4\n" );
    for( int ii = 0; ii < gpu->sim.paddedNumberOfAtoms; ii++ ){
        (void) fprintf( log, "%6d %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e %15.7e \n",
                        ii, 
                        gpu->psBornRadii->_pSysData[ii],
                        gpu->psBornForce->_pSysData[ii],
                        switchDeriviative->_pSysData[ii],

                        gpu->psGBVIData->_pSysData[ii].x,
                        gpu->psGBVIData->_pSysData[ii].y,
                        gpu->psGBVIData->_pSysData[ii].z,
                        gpu->psGBVIData->_pSysData[ii].w,

                        sigEps4->_pSysData[ii].x,
                        sigEps4->_pSysData[ii].y,
                        sigEps4->_pSysData[ii].z,
                        sigEps4->_pSysData[ii].w );

    }

}

extern __global__ void kFindBlockBoundsCutoff_kernel();
extern __global__ void kFindBlockBoundsPeriodic_kernel();

extern __global__ void kFindBlocksWithInteractionsCutoff_kernel();
extern __global__ void kFindBlocksWithInteractionsPeriodic_kernel();

extern __global__ void kFindInteractionsWithinBlocksCutoff_kernel(unsigned int*);
extern __global__ void kFindInteractionsWithinBlocksPeriodic_kernel(unsigned int*);

void kCalculateGBVISoftcoreBornSum( freeEnergyGpuContext freeEnergyGpu )
{

    unsigned int threadsPerBlock;
    static unsigned int threadsPerBlockPerMethod[3] = { 0, 0, 0 };
    static unsigned int natoms[3]                   = { 0, 0, 0 };
    gpuContext gpu                                  = freeEnergyGpu->gpuContext;
    unsigned int methodIndex                        = static_cast<unsigned int>(freeEnergyGpu->freeEnergySim.nonbondedMethod);

    if( methodIndex > 2 ){
        throw OpenMM::OpenMMException( "kCalculateGBVISoftcoreBornSum method index invalid." );
    }

    if( natoms[methodIndex] != gpu->natoms ){
        unsigned int extra                    = methodIndex == 0 ? 0 : sizeof(float);
        threadsPerBlockPerMethod[methodIndex] = std::min(getThreadsPerBlockFEP( freeEnergyGpu, (sizeof(Atom) + extra), gpu->sharedMemoryPerBlock ), gpu->sim.nonbond_threads_per_block );
        natoms[methodIndex]                   = gpu->natoms;
    }   
    threadsPerBlock = threadsPerBlockPerMethod[methodIndex];

    kClearGBVISoftcoreBornSum( gpu );
    LAUNCHERROR("kClearGBVIBornSum from kCalculateGBVISoftcoreBornSum");

    switch (freeEnergyGpu->freeEnergySim.nonbondedMethod)
    {   
        case FREE_ENERGY_NO_CUTOFF:

            if (gpu->bOutputBufferPerWarp){
                kCalculateGBVISoftcoreN2ByWarpBornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        sizeof(Atom)*threadsPerBlock>>>(gpu->sim.pWorkUnit);
            } else {
                kCalculateGBVISoftcoreN2BornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        sizeof(Atom)*threadsPerBlock>>>(gpu->sim.pWorkUnit);
            }
            break;

        case FREE_ENERGY_CUTOFF:

            kFindBlockBoundsCutoff_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsCutoff");
            kFindBlocksWithInteractionsCutoff_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsCutoff");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksCutoff_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                    sizeof(unsigned int)*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);

            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVISoftcoreCutoffByWarpBornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateGBVISoftcoreCutoffBornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit );

            break;

        case FREE_ENERGY_PERIODIC:

            kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
            LAUNCHERROR("kFindBlockBoundsPeriodic");
            kFindBlocksWithInteractionsPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
            LAUNCHERROR("kFindBlocksWithInteractionsPeriodic");
            compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, gpu->sim.pWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
            kFindInteractionsWithinBlocksPeriodic_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                    sizeof(unsigned int)*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);

            if (gpu->bOutputBufferPerWarp)
                kCalculateGBVISoftcorePeriodicByWarpBornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit );
            else
                kCalculateGBVISoftcorePeriodicBornSum_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit );

            break;

        default:
            throw OpenMM::OpenMMException( "Nonbonded softcore method not recognized." );

    }
    LAUNCHERROR("kCalculateGBVISoftcoreBornSum");

}
