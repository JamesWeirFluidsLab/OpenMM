#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "openmm/OpenMMException.h"
#include "gputypes.h"
#include "freeEnergyGpuTypes.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
using namespace std;

struct Atom {
    float x;
    float y;
    float z;
    float r;
    float sr;
    float npScale;
    float fx;
    float fy;
    float fz;
    float fb;
};

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaFreeEnergyGmxSimulation feSimDev;

extern "C"
void SetCalculateObcGbsaSoftcoreForces2Sim( freeEnergyGpuContext freeEnergyGpu )
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &freeEnergyGpu->gpuContext->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateObcGbsaSoftcoreForces2Sim copy to cSim failed");

    status = hipMemcpyToSymbol(HIP_SYMBOL( feSimDev), &freeEnergyGpu->freeEnergySim, sizeof(cudaFreeEnergyGmxSimulation));
    RTERROR(status, "hipMemcpyToSymbol: SetCalculateObcGbsaSoftcoreForces2Sim copy to feSimDev failed");

}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

// Include versions of the kernels with cutoffs.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

// Include versions of the kernels with periodic boundary conditions.

#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP
#define USE_PERIODIC
#define METHOD_NAME(a, b) a##Periodic##b
#include "kCalculateObcGbsaSoftcoreForces2.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##PeriodicByWarp##b
#include "kCalculateObcGbsaSoftcoreForces2.h"

void kCalculateObcGbsaSoftcoreForces2( freeEnergyGpuContext freeEnergyGpu )
{
    unsigned int threadsPerBlock;
    static unsigned int threadsPerBlockPerMethod[3] = { 0, 0, 0 };
    static unsigned int natoms[3]                   = { 0, 0, 0 };
    gpuContext gpu                                  = freeEnergyGpu->gpuContext;
    unsigned int methodIndex                        = static_cast<unsigned int>(freeEnergyGpu->freeEnergySim.nonbondedMethod);

    if( methodIndex > 2 ){
        throw OpenMM::OpenMMException( "kCalculateObcGbsaSoftcoreForces2 method index invalid." );
    }   

    if( natoms[methodIndex] != gpu->natoms ){
        unsigned int extra                    = methodIndex == 0 ? 0 : sizeof(float3);
        threadsPerBlockPerMethod[methodIndex] = std::min(getThreadsPerBlockFEP( freeEnergyGpu, (sizeof(Atom) + extra), gpu->sharedMemoryPerBlock ), gpu->sim.nonbond_threads_per_block );
        natoms[methodIndex]                   = gpu->natoms;
    }
    threadsPerBlock                                 = threadsPerBlockPerMethod[methodIndex];

    switch (freeEnergyGpu->freeEnergySim.nonbondedMethod)
    {
        case FREE_ENERGY_NO_CUTOFF:

            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcoreN2ByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        sizeof(Atom)*threadsPerBlock>>>(gpu->sim.pWorkUnit);
            else
                kCalculateObcGbsaSoftcoreN2Forces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        sizeof(Atom)*threadsPerBlock>>>(gpu->sim.pWorkUnit);
            break;

        case FREE_ENERGY_CUTOFF:

            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcoreCutoffByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float3))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateObcGbsaSoftcoreCutoffForces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float3))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);
            break;

        case FREE_ENERGY_PERIODIC:

            if (gpu->bOutputBufferPerWarp)
                kCalculateObcGbsaSoftcorePeriodicByWarpForces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float3))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);
            else
                kCalculateObcGbsaSoftcorePeriodicForces2_kernel<<<gpu->sim.bornForce2_blocks, threadsPerBlock,
                        (sizeof(Atom)+sizeof(float3))*threadsPerBlock>>>(gpu->sim.pInteractingWorkUnit);
            break;
    }
    LAUNCHERROR("kCalculateObcGbsaSoftcoreForces2");
}
