#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "cudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"
#include "kCalculateAmoebaCudaVdwParticle.h"
#include "amoebaScaleFactors.h"

#include <stdio.h>

extern int isNanOrInfinity( double number );

using namespace std;

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaCudaVdw14_7Sim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaVdw14_7Sim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaVdw14_7FieldSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaCudaVdw14_7Sim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaVdw14_7Sim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaVdw14_7Sim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

__device__ void zeroVdw14_7SharedForce( struct Vdw14_7Particle* sA ) 
{
    // zero shared fields

    sA->force[0]              = 0.0f;
    sA->force[1]              = 0.0f;
    sA->force[2]              = 0.0f;

    sA->tempForce[0]          = 0.0f;
    sA->tempForce[1]          = 0.0f;
    sA->tempForce[2]          = 0.0f;

}

__device__ void loadVdw14_7Shared( struct Vdw14_7Particle* sA, unsigned int atomI,
                                   float4* atomCoord, float2* vdwParameters )
{
    // coordinates, sigma, epsilon

    sA->x                        = atomCoord[atomI].x;
    sA->y                        = atomCoord[atomI].y;
    sA->z                        = atomCoord[atomI].z;

    sA->sigma                    = vdwParameters[atomI].x;
    sA->epsilon                  = vdwParameters[atomI].y;

}

__device__ void getVdw14_7CombindedSigmaEpsilon_kernel( int sigmaCombiningRule, float iSigma, float jSigma, float* combindedSigma,
                                                        int epsilonCombiningRule, float iEpsilon, float jEpsilon, float* combindedEpsilon )
{
    if( sigmaCombiningRule == 1 ){
        *combindedSigma      = iSigma + jSigma;
    } else if( sigmaCombiningRule == 2 ){
        *combindedSigma      = 2.0f*sqrtf( iSigma*jSigma );
    } else {
        float iSigma2        = iSigma*iSigma;
        float jSigma2        = jSigma*jSigma;
        *combindedSigma      = 2.0f*( iSigma2*iSigma + jSigma2*jSigma )/( iSigma2 + jSigma2 );
    }

    if( epsilonCombiningRule == 1 ){
        *combindedEpsilon    = iEpsilon + jEpsilon;
    } else if( epsilonCombiningRule == 2 ){
        *combindedEpsilon    = 2.0f*sqrtf( iEpsilon*jEpsilon );
    } else if( epsilonCombiningRule == 3 ){
        float iEpsilon2      = iEpsilon*iEpsilon;
        float jEpsilon2      = jEpsilon*jEpsilon;
        *combindedEpsilon    = 2.0f*( iEpsilon2*iEpsilon + jEpsilon2*jEpsilon )/( iEpsilon2 + jEpsilon2 );
    } else {
        float epsilonS       = sqrtf( iEpsilon ) + sqrtf( jEpsilon );
        *combindedEpsilon    = 4.0f*( iEpsilon*jEpsilon )/( epsilonS*epsilonS );
    }   

}

// lookup table w/ linear interpolation

__device__ void lookupLinearVdwTaper( float r, float* taper, float* dtaper )
{
    if( r > (cAmoebaSim.vdwCutoff - cAmoebaSim.vdwTaperDelta) ){
        *taper = *dtaper = 0.0f;
    } else {

        int index        = (int) (floor( (r - cAmoebaSim.vdwTaperCutoff)/cAmoebaSim.vdwTaperDelta));
//        int index        = (int) round( (r - cAmoebaSim.vdwTaperCutoff)/cAmoebaSim.vdwTaperDelta);
        float slope      = (cAmoebaSim.vdwTaperTable[index+1] - cAmoebaSim.vdwTaperTable[index])/cAmoebaSim.vdwTaperDelta;
        float intercept  =  cAmoebaSim.vdwTaperTable[index+1] - slope*(cAmoebaSim.vdwTaperDelta*static_cast<float>(index+1));
        *taper           = slope*(r-cAmoebaSim.vdwTaperCutoff) + intercept; 

        slope            = (cAmoebaSim.vdw_dTaperTable[index+1] - cAmoebaSim.vdw_dTaperTable[index])/cAmoebaSim.vdwTaperDelta;
        intercept        =  cAmoebaSim.vdw_dTaperTable[index+1]  - slope*(cAmoebaSim.vdwTaperDelta*static_cast<float>(index+1));
        *dtaper          = slope*(r-cAmoebaSim.vdwTaperCutoff) + intercept; 
    }
}

// lookup table w/ quadratic interpolation

__device__ void lookupVdwTaper( float r, float* taper, float* dtaper )
{
    if( r > (cAmoebaSim.vdwCutoff - 2.0f*cAmoebaSim.vdwTaperDelta) ){
        *taper = *dtaper = 0.0f;
    } else {

        float x          = r - cAmoebaSim.vdwTaperCutoff;
//        int index        = (int) (floor(x)/cAmoebaSim.vdwTaperDelta);
        int index        = (int) round(x/cAmoebaSim.vdwTaperDelta);
        if( index ){

            float x0         = cAmoebaSim.vdwTaperDelta*static_cast<float>(index-1);
            float y0         = cAmoebaSim.vdwTaperTable[index-1];
    
            float x1         = x0 + cAmoebaSim.vdwTaperDelta;
            float y1         = cAmoebaSim.vdwTaperTable[index];
    
            float x2         = x1 + cAmoebaSim.vdwTaperDelta;
            float y2         = cAmoebaSim.vdwTaperTable[index+1];
    
            *taper           = y0*( (x-x1)*(x-x2)/((x0-x1)*(x0-x2))) + 
                               y1*( (x-x0)*(x-x2)/((x1-x0)*(x1-x2))) + 
                               y2*( (x-x0)*(x-x1)/((x2-x0)*(x2-x1)));
    
                  y0         = cAmoebaSim.vdw_dTaperTable[index-1];
                  y1         = cAmoebaSim.vdw_dTaperTable[index];
                  y2         = cAmoebaSim.vdw_dTaperTable[index+1];
    
            *dtaper          = y0*( (x-x1)*(x-x2)/((x0-x1)*(x0-x2))) + 
                               y1*( (x-x0)*(x-x2)/((x1-x0)*(x1-x2))) + 
                               y2*( (x-x0)*(x-x1)/((x2-x0)*(x2-x1)));
    
        } else {
            *taper  = 1.0f;
            *dtaper = 0.0f;
        }
    }
}

__device__ void calculateVdw14_7PairIxn_kernel( float combindedSigma,    float combindedEpsilon,
                                                float force[3], float* energy)
{

    const float deltaHalM1 = 0.07f;
    const float deltaHal   = 1.07f;
    const float gammaHal   = 1.12f;
    const float gammaHalM1 = 0.12f;

    // ---------------------------------------------------------------------------------------
    
    // on input force[i] is assummed to contain delta[i] for coordinates of atom I and J 
    
    float r2                                     = force[0]*force[0] + force[1]*force[1] + force[2]*force[2];
    if( r2 > cAmoebaSim.vdwCutoff2 ){
        *energy = force[0] = force[1] = force[2] = 0.0f;
        return;
    }
    float rI                                     =  rsqrtf( r2 );
    float r                                      =  1.0f/rI;
    float r6                                     =  r2*r2*r2;
    float r7                                     =  r6*r;
 
    float combindedSigma7                        = combindedSigma*combindedSigma;
    combindedSigma7                              = combindedSigma7*combindedSigma7*combindedSigma7*combindedSigma;

    float rho                                    = r7 + combindedSigma7*gammaHalM1;
    float rhoInverse                             = 1.0f/rho;
 
    float tau                                    = deltaHal/(r + deltaHalM1*combindedSigma);
    float tau7                                   = tau*tau*tau;
         tau7                                    = tau7*tau7*tau;
    float dTau                                   = tau/deltaHal;
    
    float tmp                                    = combindedSigma7*rhoInverse;
    float gTau                                   = combindedEpsilon*tau7*r6*gammaHal*tmp*tmp;
 
    *energy                                      = combindedEpsilon*combindedSigma7*tau7*( (combindedSigma7*gammaHal*rhoInverse) - 2.0f);
    float deltaE                                 = (-7.0f*(dTau*(*energy) + gTau))*rI;
 
    if( r > cAmoebaSim.vdwTaperCutoff ){ 

        float taper, dtaper;
        lookupVdwTaper( r, &taper, &dtaper );
        //lookupLinearVdwTaper( r, &taper, &dtaper );
        deltaE   = (*energy)*dtaper + deltaE*taper;
        *energy *= taper;
    }

    force[0]                                    *= deltaE;
    force[1]                                    *= deltaE;
    force[2]                                    *= deltaE;


}

// perform reduction of force on H's and add to heavy atom partner
// input force is the Vdw force
// output force is the cumulative force

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_NONBOND_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_NONBOND_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_NONBOND_THREADS_PER_BLOCK, 1)
#endif
void kCalculateAmoebaVdw14_7Reduction_kernel( float* inputForce, float4* outputForce )
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    while (pos < cAmoebaSim.amoebaVdwReductions )
    {
        int4   atomIndices              = cAmoebaSim.pAmoebaVdwReductionID[pos];
        float4 forceA;
        float4 forceB;
        float4 forceC;
        float4 forceD;

        int index                       = 3*atomIndices.x;
        forceA.x                        = inputForce[index];
        forceA.y                        = inputForce[index+1];
        forceA.z                        = inputForce[index+2];

        index                           = 3*atomIndices.y;
        forceB.x                        = inputForce[index];
        forceB.y                        = inputForce[index+1];
        forceB.z                        = inputForce[index+2];

        index                           = 3*atomIndices.z;
        forceC.x                        = inputForce[index];
        forceC.y                        = inputForce[index+1];
        forceC.z                        = inputForce[index+2];

        index                           = 3*atomIndices.w;
        forceD.x                        = inputForce[index];
        forceD.y                        = inputForce[index+1];
        forceD.z                        = inputForce[index+2];

        float  reductionFactor          = cAmoebaSim.pAmoebaVdwReduction[pos];
        float  reductionFactorM1        = 1.0f - reductionFactor;
        
        float4 forceTemp1;
        forceTemp1.x                    = reductionFactor*forceB.x;
        forceTemp1.y                    = reductionFactor*forceB.y;
        forceTemp1.z                    = reductionFactor*forceB.z;

        forceA.x                       += reductionFactorM1*forceB.x;
        forceA.y                       += reductionFactorM1*forceB.y;
        forceA.z                       += reductionFactorM1*forceB.z;

        outputForce[atomIndices.y].x   += forceTemp1.x;
        outputForce[atomIndices.y].y   += forceTemp1.y;
        outputForce[atomIndices.y].z   += forceTemp1.z;

        reductionFactor                 = atomIndices.x != atomIndices.z ? reductionFactor   : 0.0f;
        reductionFactorM1               = atomIndices.x != atomIndices.z ? reductionFactorM1 : 0.0f;

        forceTemp1.x                    = reductionFactor*forceC.x;
        forceTemp1.y                    = reductionFactor*forceC.y;
        forceTemp1.z                    = reductionFactor*forceC.z;

        forceA.x                       += reductionFactorM1*forceC.x;
        forceA.y                       += reductionFactorM1*forceC.y;
        forceA.z                       += reductionFactorM1*forceC.z;

        outputForce[atomIndices.z].x   += forceTemp1.x;
        outputForce[atomIndices.z].y   += forceTemp1.y;
        outputForce[atomIndices.z].z   += forceTemp1.z;

        reductionFactor                 = atomIndices.x != atomIndices.w ? reductionFactor   : 0.0f;
        reductionFactorM1               = atomIndices.x != atomIndices.w ? reductionFactorM1 : 0.0f;

        forceTemp1.x                    = reductionFactor*forceD.x;
        forceTemp1.y                    = reductionFactor*forceD.y;
        forceTemp1.z                    = reductionFactor*forceD.z;

        forceA.x                       += reductionFactorM1*forceD.x;
        forceA.y                       += reductionFactorM1*forceD.y;
        forceA.z                       += reductionFactorM1*forceD.z;

        outputForce[atomIndices.w].x   += forceTemp1.x;
        outputForce[atomIndices.w].y   += forceTemp1.y;
        outputForce[atomIndices.w].z   += forceTemp1.z;

        outputForce[atomIndices.x].x   += forceA.x;
        outputForce[atomIndices.x].y   += forceA.y;
        outputForce[atomIndices.x].z   += forceA.z;
        
        pos                            += blockDim.x * gridDim.x;
    }
}

__device__ void sumTempBuffer( Vdw14_7Particle& atomI, Vdw14_7Particle& atomJ ){

    atomI.tempForce[0]  += atomJ.tempForce[0];
    atomI.tempForce[1]  += atomJ.tempForce[1];
    atomI.tempForce[2]  += atomJ.tempForce[2];
}

static void kCalculateAmoebaVdw14_7Reduction(amoebaGpuContext amoebaGpu, CUDAStream<float>* vdwOutputArray, CUDAStream<float4>* forceOutputArray )
{
    unsigned int threadsPerBlock;
    if (amoebaGpu->gpuContext->sm_version >= SM_20)
        threadsPerBlock = GF1XX_NONBOND_THREADS_PER_BLOCK;
    else if (amoebaGpu->gpuContext->sm_version >= SM_12)
        threadsPerBlock = GT2XX_NONBOND_THREADS_PER_BLOCK;
    else
        threadsPerBlock = G8X_NONBOND_THREADS_PER_BLOCK;
    kCalculateAmoebaVdw14_7Reduction_kernel<<<amoebaGpu->gpuContext->sim.blocks, threadsPerBlock>>>(
                               vdwOutputArray->_pDevData, forceOutputArray->_pDevData );
    LAUNCHERROR("kCalculateAmoebaVdw14_7Reduction");
}

// perform reduction of coordinate on H's and add to heavy atom partner
// input coordinate is the Vdw coordinate
// output coordinate is the cumulative coordinate

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kCalculateAmoebaVdw14_7CoordinateReduction_kernel( float4* inputCoordinate, float4* outputCoordinate )
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    while (pos < cAmoebaSim.amoebaVdwReductions )
    {
        int4   atomIndices              = cAmoebaSim.pAmoebaVdwReductionID[pos];
        float4 coordinateA;
        float4 coordinateB;
        float4 coordinateC;
        float4 coordinateD;

        int index                             = atomIndices.x;
        coordinateA.x                         = inputCoordinate[index].x;
        coordinateA.y                         = inputCoordinate[index].y;
        coordinateA.z                         = inputCoordinate[index].z;

        index                                 = atomIndices.y;
        coordinateB.x                         = inputCoordinate[index].x;
        coordinateB.y                         = inputCoordinate[index].y;
        coordinateB.z                         = inputCoordinate[index].z;

        index                                 = atomIndices.z;
        coordinateC.x                         = inputCoordinate[index].x;
        coordinateC.y                         = inputCoordinate[index].y;
        coordinateC.z                         = inputCoordinate[index].z;

        index                                 = atomIndices.w;
        coordinateD.x                         = inputCoordinate[index].x;
        coordinateD.y                         = inputCoordinate[index].y;
        coordinateD.z                         = inputCoordinate[index].z;

        float  reductionFactor                = cAmoebaSim.pAmoebaVdwReduction[pos];
        float  reductionFactorM1              = 1.0f - reductionFactor;
        
        float4 coordinateTemp1;
        coordinateTemp1.x                     = reductionFactor*coordinateB.x + reductionFactorM1*coordinateA.x;
        coordinateTemp1.y                     = reductionFactor*coordinateB.y + reductionFactorM1*coordinateA.y;
        coordinateTemp1.z                     = reductionFactor*coordinateB.z + reductionFactorM1*coordinateA.z;

        outputCoordinate[atomIndices.y].x     = coordinateTemp1.x;
        outputCoordinate[atomIndices.y].y     = coordinateTemp1.y;
        outputCoordinate[atomIndices.y].z     = coordinateTemp1.z;

        reductionFactor                       = atomIndices.x != atomIndices.z ? reductionFactor   : 1.0f;
        reductionFactorM1                     = atomIndices.x != atomIndices.z ? reductionFactorM1 : 0.0f;

        coordinateTemp1.x                     = reductionFactor*coordinateC.x + reductionFactorM1*coordinateA.x;
        coordinateTemp1.y                     = reductionFactor*coordinateC.y + reductionFactorM1*coordinateA.y;
        coordinateTemp1.z                     = reductionFactor*coordinateC.z + reductionFactorM1*coordinateA.z;

        outputCoordinate[atomIndices.z].x     = coordinateTemp1.x;
        outputCoordinate[atomIndices.z].y     = coordinateTemp1.y;
        outputCoordinate[atomIndices.z].z     = coordinateTemp1.z;

        reductionFactor                       = atomIndices.x != atomIndices.w ? reductionFactor   : 1.0f;
        reductionFactorM1                     = atomIndices.x != atomIndices.w ? reductionFactorM1 : 0.0f;

        coordinateTemp1.x                     = reductionFactor*coordinateD.x + reductionFactorM1*coordinateA.x;
        coordinateTemp1.y                     = reductionFactor*coordinateD.y + reductionFactorM1*coordinateA.y;
        coordinateTemp1.z                     = reductionFactor*coordinateD.z + reductionFactorM1*coordinateA.z;

        outputCoordinate[atomIndices.w].x     = coordinateTemp1.x;
        outputCoordinate[atomIndices.w].y     = coordinateTemp1.y;
        outputCoordinate[atomIndices.w].z     = coordinateTemp1.z;

        pos                                  += blockDim.x * gridDim.x;
    }
}

static void kCalculateAmoebaVdw14_7CoordinateReduction(amoebaGpuContext amoebaGpu,
                                                       CUDAStream<float4>* coordinateArray,
                                                       CUDAStream<float4>* reducedCoordinateArray)
{
    unsigned int threadsPerBlock;
    if (amoebaGpu->gpuContext->sm_version >= SM_20)
        threadsPerBlock = GF1XX_THREADS_PER_BLOCK;
    else if (amoebaGpu->gpuContext->sm_version >= SM_12)
        threadsPerBlock = GT2XX_THREADS_PER_BLOCK;
    else
        threadsPerBlock = G8X_THREADS_PER_BLOCK;
    kCalculateAmoebaVdw14_7CoordinateReduction_kernel<<<amoebaGpu->gpuContext->sim.blocks, threadsPerBlock>>>(
                               coordinateArray->_pDevData, reducedCoordinateArray->_pDevData );
    LAUNCHERROR("kCalculateAmoebaVdw14_7CoordinateReduction");
}

// perform reduction of force on H's and add to heavy atom partner
// input force is the Vdw force
// output force is the cumulative force

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kCalculateAmoebaVdw14_7NonReduction_kernel( float* inputForce, float4* outputForce )
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    while (pos < cAmoebaSim.amoebaVdwNonReductions )
    {
        int   atomIndex             = cAmoebaSim.pAmoebaVdwNonReductionID[pos];
        int index                   = 3*atomIndex;
        outputForce[atomIndex].x   += inputForce[index];
        outputForce[atomIndex].y   += inputForce[index+1];
        outputForce[atomIndex].z   += inputForce[index+2];
        
        pos                        += blockDim.x * gridDim.x;
    }
}

static void kCalculateAmoebaVdw14_7NonReduction(amoebaGpuContext amoebaGpu, CUDAStream<float>* vdwOutputArray, CUDAStream<float4>* forceOutputArray )
{
    kCalculateAmoebaVdw14_7NonReduction_kernel<<<amoebaGpu->gpuContext->sim.blocks, amoebaGpu->gpuContext->sim.threads_per_block>>>(
                               vdwOutputArray->_pDevData, forceOutputArray->_pDevData );
    LAUNCHERROR("kCalculateAmoebaVdw14_7MonReduction");
}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaVdw14_7.h"

#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaVdw14_7.h"
#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP 

#define USE_CUTOFF
#define METHOD_NAME(a, b) a##Cutoff##b
#include "kCalculateAmoebaCudaVdw14_7.h"
#undef METHOD_NAME

#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##CutoffByWarp##b
#include "kCalculateAmoebaCudaVdw14_7.h"
#undef METHOD_NAME
#undef USE_OUTPUT_BUFFER_PER_WARP 
#undef USE_CUTOFF

// reduce psWorkArray_3_1 -> outputArray

static void kReduceVdw14_7(amoebaGpuContext amoebaGpu, CUDAStream<float>* outputArray )
{
    gpuContext gpu = amoebaGpu->gpuContext;
    kReduceFields_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.bsf_reduce_threads_per_block>>>(
                           gpu->sim.paddedNumberOfAtoms*3, gpu->sim.outputBuffers,
                           amoebaGpu->psWorkArray_3_1->_pDevData, outputArray->_pDevData, 0 );
    LAUNCHERROR("kReduceVdw14_7");
}

__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(GF1XX_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(GT2XX_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(G8X_THREADS_PER_BLOCK, 1)
#endif
void kCalculateAmoebaVdw14_7CopyCoordinates_kernel( unsigned int bufferLength, float4* toCopy, float4* copy )
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    while (pos < bufferLength )
    {   
        copy[pos].x       = toCopy[pos].x;
        copy[pos].y       = toCopy[pos].y;
        copy[pos].z       = toCopy[pos].z;
        copy[pos].w       = toCopy[pos].w;
        pos              += gridDim.x * blockDim.x;
    }   
}

void kCalculateAmoebaVdw14_7CopyCoordinates( amoebaGpuContext amoebaGpu, CUDAStream<float4>* toCopy, CUDAStream<float4>* copy )
{
    kCalculateAmoebaVdw14_7CopyCoordinates_kernel<<<amoebaGpu->gpuContext->blocksPerSM, amoebaGpu->gpuContext->sim.threads_per_block>>>( amoebaGpu->gpuContext->sim.paddedNumberOfAtoms, 
                      toCopy->_pDevData, copy->_pDevData );
    LAUNCHERROR("kCalculateAmoebaVdw14_7CopyCoordinates");
}

/**---------------------------------------------------------------------------------------

   Compute Vdw 14-7

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

void kCalculateAmoebaVdw14_7Forces( amoebaGpuContext amoebaGpu, int applyCutoff )
{
  
   // ---------------------------------------------------------------------------------------

    static int threadsPerBlock    = 0;

   // ---------------------------------------------------------------------------------------

     gpuContext gpu    = amoebaGpu->gpuContext;

    // set threads/block first time through

    // on first pass, set threads/block

    if( threadsPerBlock == 0 ){ 
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 384; 
        else if (gpu->sm_version >= SM_12)
            maxThreads = 192; 
        else
            maxThreads = 128;
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(Vdw14_7Particle), gpu->sharedMemoryPerBlock ), maxThreads);
    }    


    kCalculateAmoebaVdw14_7CopyCoordinates( amoebaGpu, gpu->psPosq4, amoebaGpu->psAmoebaVdwCoordinates );
    kCalculateAmoebaVdw14_7CoordinateReduction( amoebaGpu, amoebaGpu->psAmoebaVdwCoordinates, amoebaGpu->psAmoebaVdwCoordinates );

    // clear output arrays

    kClearFields_3( amoebaGpu, 1 );

    if( applyCutoff ){

        kFindBlockBoundsPeriodic_kernel<<<(gpu->psGridBoundingBox->_length+63)/64, 64>>>();
        LAUNCHERROR("kFindBlockBoundsVdwPeriodic");
        kFindBlocksWithInteractionsVdwPeriodic_kernel<<<gpu->sim.interaction_blocks, gpu->sim.interaction_threads_per_block>>>();
        LAUNCHERROR("kFindBlocksWithInteractionsVdwPeriodic");
        compactStream(gpu->compactPlan, gpu->sim.pInteractingWorkUnit, amoebaGpu->amoebaSim.pVdwWorkUnit, gpu->sim.pInteractionFlag, gpu->sim.workUnits, gpu->sim.pInteractionCount);
        kFindInteractionsWithinBlocksVdwPeriodic_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.nonbond_threads_per_block,
                    sizeof(unsigned int)*gpu->sim.nonbond_threads_per_block>>>(gpu->sim.pInteractingWorkUnit);
        LAUNCHERROR("kFindInteractionsWithinBlocksVdwPeriodic");

        if (gpu->bOutputBufferPerWarp){
            kCalculateAmoebaVdw14_7CutoffByWarp_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(Vdw14_7Particle)*threadsPerBlock>>>(
                                                                gpu->sim.pInteractingWorkUnit,
                                                                amoebaGpu->psAmoebaVdwCoordinates->_pDevData,
                                                                amoebaGpu->psVdwSigmaEpsilon->_pDevData,
                                                                amoebaGpu->vdwSigmaCombiningRule,
                                                                amoebaGpu->vdwEpsilonCombiningRule,
                                                                amoebaGpu->psWorkArray_3_1->_pDevData );
        } else {

            kCalculateAmoebaVdw14_7Cutoff_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(Vdw14_7Particle)*threadsPerBlock>>>(
                                                                gpu->sim.pInteractingWorkUnit,
                                                                amoebaGpu->psAmoebaVdwCoordinates->_pDevData,
                                                                amoebaGpu->psVdwSigmaEpsilon->_pDevData,
                                                                amoebaGpu->vdwSigmaCombiningRule,
                                                                amoebaGpu->vdwEpsilonCombiningRule,
                                                                amoebaGpu->psWorkArray_3_1->_pDevData );
        }
        LAUNCHERROR("kCalculateAmoebaVdw14_7Cutoff");  

    } else {

        if (gpu->bOutputBufferPerWarp){

            kCalculateAmoebaVdw14_7N2ByWarp_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(Vdw14_7Particle)*threadsPerBlock>>>(
                                                                amoebaGpu->psVdwWorkUnit->_pDevData,
                                                                amoebaGpu->psAmoebaVdwCoordinates->_pDevData,
                                                                amoebaGpu->psVdwSigmaEpsilon->_pDevData,
                                                                amoebaGpu->vdwSigmaCombiningRule,
                                                                amoebaGpu->vdwEpsilonCombiningRule,
                                                                amoebaGpu->psWorkArray_3_1->_pDevData );
        } else {

            kCalculateAmoebaVdw14_7N2_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(Vdw14_7Particle)*threadsPerBlock>>>(
                                                                amoebaGpu->psVdwWorkUnit->_pDevData,
                                                                amoebaGpu->psAmoebaVdwCoordinates->_pDevData,
                                                                amoebaGpu->psVdwSigmaEpsilon->_pDevData,
                                                                amoebaGpu->vdwSigmaCombiningRule,
                                                                amoebaGpu->vdwEpsilonCombiningRule,
                                                                amoebaGpu->psWorkArray_3_1->_pDevData );
    
        }
        LAUNCHERROR("kCalculateAmoebaVdw14_7N2");  
    }

    kReduceVdw14_7( amoebaGpu, amoebaGpu->psWorkArray_3_2 );

    kCalculateAmoebaVdw14_7Reduction( amoebaGpu, amoebaGpu->psWorkArray_3_2, amoebaGpu->gpuContext->psForce4 );
    kCalculateAmoebaVdw14_7NonReduction( amoebaGpu, amoebaGpu->psWorkArray_3_2, amoebaGpu->gpuContext->psForce4 );

}
