#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "amoebaGpuTypes.h"
#include "cudaKernels.h"
#include "amoebaCudaKernels.h"
#include "bbsort.h"
#include <sstream>

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

/* Cuda compiler on Windows does not recognized "static const float" values */
#define LOCAL_HACK_PI 3.1415926535897932384626433832795f

void SetCalculateAmoebaPMESim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));
    RTERROR(status, "SetCalculateAmoebaPMESim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));
    RTERROR(status, "SetCalculateAmoebaPMESim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

#define ARRAY(x,y) array[(x)-1+((y)-1)*AMOEBA_PME_ORDER]

/**
 * This is called from computeBsplines().  It calculates the spline coefficients for a single atom along a single axis.
 */
__device__ void computeBSplinePoint(float4* thetai, float w, float* array)
{
    // initialization to get to 2nd order recursion

    ARRAY(2,2) = w;
    ARRAY(2,1) = 1.0f - w;

    // perform one pass to get to 3rd order recursion

    ARRAY(3,3) = 0.5f * w * ARRAY(2,2);
    ARRAY(3,2) = 0.5f * ((1.0f+w)*ARRAY(2,1)+(2.0f-w)*ARRAY(2,2));
    ARRAY(3,1) = 0.5f * (1.0f-w) * ARRAY(2,1);

    // compute standard B-spline recursion to desired order

    for (int i = 4; i <= AMOEBA_PME_ORDER; i++)
    {
        int k = i - 1;
        float denom = 1.0f / k;
        ARRAY(i,i) = denom * w * ARRAY(k,k);
        for (int j = 1; j <= i-2; j++)
            ARRAY(i,i-j) = denom * ((w+j)*ARRAY(k,i-j-1)+(i-j-w)*ARRAY(k,i-j));
        ARRAY(i,1) = denom * (1.0f-w) * ARRAY(k,1);
    }

    // get coefficients for the B-spline first derivative

    int k = AMOEBA_PME_ORDER - 1;
    ARRAY(k,AMOEBA_PME_ORDER) = ARRAY(k,AMOEBA_PME_ORDER-1);
    for (int i = AMOEBA_PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // get coefficients for the B-spline second derivative

    k = AMOEBA_PME_ORDER - 2;
    ARRAY(k,AMOEBA_PME_ORDER-1) = ARRAY(k,AMOEBA_PME_ORDER-2);
    for (int i = AMOEBA_PME_ORDER-2; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,AMOEBA_PME_ORDER) = ARRAY(k,AMOEBA_PME_ORDER-1);
    for (int i = AMOEBA_PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // get coefficients for the B-spline third derivative

    k = AMOEBA_PME_ORDER - 3;
    ARRAY(k,AMOEBA_PME_ORDER-2) = ARRAY(k,AMOEBA_PME_ORDER-3);
    for (int i = AMOEBA_PME_ORDER-3; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,AMOEBA_PME_ORDER-1) = ARRAY(k,AMOEBA_PME_ORDER-2);
    for (int i = AMOEBA_PME_ORDER-2; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);
    ARRAY(k,AMOEBA_PME_ORDER) = ARRAY(k,AMOEBA_PME_ORDER-1);
    for (int i = AMOEBA_PME_ORDER-1; i >= 2; i--)
        ARRAY(k,i) = ARRAY(k,i-1) - ARRAY(k,i);
    ARRAY(k,1) = -ARRAY(k,1);

    // copy coefficients from temporary to permanent storage

    for (int i = 1; i <= AMOEBA_PME_ORDER; i++)
        thetai[i-1] = make_float4(ARRAY(AMOEBA_PME_ORDER,i), ARRAY(AMOEBA_PME_ORDER-1,i), ARRAY(AMOEBA_PME_ORDER-2,i), ARRAY(AMOEBA_PME_ORDER-3,i));
}

/**
 * Compute bspline coefficients.
 */
__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(448, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(160, 1)
#else
__launch_bounds__(160, 1)
#endif
void kComputeAmoebaBsplines_kernel()
{
    extern __shared__ float bsplines_cache[]; // size = block_size*pme_order*pme_order
    float* array = &bsplines_cache[threadIdx.x*AMOEBA_PME_ORDER*AMOEBA_PME_ORDER];

    //  get the B-spline coefficients for each multipole site

    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < cSim.atoms; i += blockDim.x*gridDim.x) {
        float4 posq = cSim.pPosq[i];
        posq.x -= floorf(posq.x*cSim.invPeriodicBoxSizeX)*cSim.periodicBoxSizeX;
        posq.y -= floorf(posq.y*cSim.invPeriodicBoxSizeY)*cSim.periodicBoxSizeY;
        posq.z -= floorf(posq.z*cSim.invPeriodicBoxSizeZ)*cSim.periodicBoxSizeZ;

        // First axis.

        float w = posq.x*cSim.invPeriodicBoxSizeX;
        float fr = cSim.pmeGridSize.x*(w-(int)(w+0.5f)+0.5f);
        int ifr = (int) fr;
        w = fr - ifr;
        int igrid1 = ifr-AMOEBA_PME_ORDER+1;
        computeBSplinePoint(&cAmoebaSim.pThetai1[i*AMOEBA_PME_ORDER], w, array);

        // Second axis.

        w = posq.y*cSim.invPeriodicBoxSizeY;
        fr = cSim.pmeGridSize.y*(w-(int)(w+0.5f)+0.5f);
        ifr = (int) fr;
        w = fr - ifr;
        int igrid2 = ifr-AMOEBA_PME_ORDER+1;
        computeBSplinePoint(&cAmoebaSim.pThetai2[i*AMOEBA_PME_ORDER], w, array);

        // Third axis.

        w = posq.z*cSim.invPeriodicBoxSizeZ;
        fr = cSim.pmeGridSize.z*(w-(int)(w+0.5f)+0.5f);
        ifr = (int) fr;
        w = fr - ifr;
        int igrid3 = ifr-AMOEBA_PME_ORDER+1;
        computeBSplinePoint(&cAmoebaSim.pThetai3[i*AMOEBA_PME_ORDER], w, array);

        // Record the grid point.

        igrid1 += (igrid1 < 0 ? cSim.pmeGridSize.x : 0);
        igrid2 += (igrid2 < 0 ? cSim.pmeGridSize.y : 0);
        igrid3 += (igrid3 < 0 ? cSim.pmeGridSize.z : 0);
        cAmoebaSim.pIgrid[i] = make_int4(igrid1, igrid2, igrid3, 0);
        cSim.pPmeAtomGridIndex[i] = make_int2(i, igrid1*cSim.pmeGridSize.y*cSim.pmeGridSize.z+igrid2*cSim.pmeGridSize.z+igrid3);
    }
}

/**
 * For each grid point, find the range of sorted atoms associated with that point.
 */
__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(1024, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(512, 1)
#else
__launch_bounds__(256, 1)
#endif
void kFindAmoebaAtomRangeForGrid_kernel()
{
    int thread = blockIdx.x*blockDim.x+threadIdx.x;
    int start = (cSim.atoms*thread)/(blockDim.x*gridDim.x);
    int end = (cSim.atoms*(thread+1))/(blockDim.x*gridDim.x);
    int last = (start == 0 ? -1 : cSim.pPmeAtomGridIndex[start-1].y);
    for (int i = start; i < end; ++i)
    {
        int2 atomData = cSim.pPmeAtomGridIndex[i];
        int gridIndex = atomData.y;
        if (gridIndex != last)
        {
            for (int j = last+1; j <= gridIndex; ++j)
                cSim.pPmeAtomRange[j] = i;
            last = gridIndex;
        }

        // The grid index won't be needed again.  Reuse that component to hold the z index, thus saving
        // some work in the charge spreading kernel.

        float posz = cSim.pPosq[atomData.x].z;
        posz -= floorf(posz*cSim.invPeriodicBoxSizeZ)*cSim.periodicBoxSizeZ;
        float w = posz*cSim.invPeriodicBoxSizeZ;
        float fr = cSim.pmeGridSize.z*(w-(int)(w+0.5f)+0.5f);
        int z = ((int) fr)-AMOEBA_PME_ORDER+1;
        cSim.pPmeAtomGridIndex[i].y = z;
    }

    // Fill in values beyond the last atom.

    if (thread == blockDim.x*gridDim.x-1)
    {
        int gridSize = cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
        for (int j = last+1; j <= gridSize; ++j)
            cSim.pPmeAtomRange[j] = cSim.atoms;
    }
}
__global__
__launch_bounds__(64, 10)
void kGridSpreadFixedMultipoles_kernel()
{
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    unsigned int numGridPoints = cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
    unsigned int numThreads = gridDim.x*blockDim.x;
    for (int gridIndex = blockIdx.x*blockDim.x+threadIdx.x; gridIndex < numGridPoints; gridIndex += numThreads)
    {
        int3 gridPoint;
        gridPoint.x = gridIndex/(cSim.pmeGridSize.y*cSim.pmeGridSize.z);
        int remainder = gridIndex-gridPoint.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
        gridPoint.y = remainder/cSim.pmeGridSize.z;
        gridPoint.z = remainder-gridPoint.y*cSim.pmeGridSize.z;
        float result = 0.0f;
        for (int ix = 0; ix < AMOEBA_PME_ORDER; ++ix)
        {
            int x = gridPoint.x-ix+(gridPoint.x >= ix ? 0 : cSim.pmeGridSize.x);
            for (int iy = 0; iy < AMOEBA_PME_ORDER; ++iy)
            {
                int y = gridPoint.y-iy+(gridPoint.y >= iy ? 0 : cSim.pmeGridSize.y);
                int z1 = gridPoint.z-AMOEBA_PME_ORDER+1;
                z1 += (z1 >= 0 ? 0 : cSim.pmeGridSize.z);
                int z2 = (z1 < gridPoint.z ? gridPoint.z : cSim.pmeGridSize.z-1);
                int gridIndex1 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+z1;
                int gridIndex2 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+z2;
                int firstAtom = cSim.pPmeAtomRange[gridIndex1];
                int lastAtom = cSim.pPmeAtomRange[gridIndex2+1];
                for (int i = firstAtom; i < lastAtom; ++i)
                {
                    int2 atomData = cSim.pPmeAtomGridIndex[i];
                    int atomIndex = atomData.x;
                    int z = atomData.y;
                    int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : cSim.pmeGridSize.z);
                    if( iz >= cSim.pmeGridSize.z ){
                        iz -= cSim.pmeGridSize.z;
                    }
                    float atomCharge = cSim.pPosq[atomIndex].w;
                    float atomDipoleX = xscale*cAmoebaSim.pLabFrameDipole[atomIndex*3];
                    float atomDipoleY = yscale*cAmoebaSim.pLabFrameDipole[atomIndex*3+1];
                    float atomDipoleZ = zscale*cAmoebaSim.pLabFrameDipole[atomIndex*3+2];
                    float atomQuadrupoleXX = xscale*xscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9];
                    float atomQuadrupoleXY = 2*xscale*yscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+1];
                    float atomQuadrupoleXZ = 2*xscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+2];
                    float atomQuadrupoleYY = yscale*yscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+4];
                    float atomQuadrupoleYZ = 2*yscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+5];
                    float atomQuadrupoleZZ = zscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+8];
                    float4 t = cAmoebaSim.pThetai1[atomIndex*AMOEBA_PME_ORDER+ix];
                    float4 u = cAmoebaSim.pThetai2[atomIndex*AMOEBA_PME_ORDER+iy];
                    float4 v = cAmoebaSim.pThetai3[atomIndex*AMOEBA_PME_ORDER+iz];
                    float term0 = atomCharge*u.x*v.x + atomDipoleY*u.y*v.x + atomDipoleZ*u.x*v.y + atomQuadrupoleYY*u.z*v.x + atomQuadrupoleZZ*u.x*v.z + atomQuadrupoleYZ*u.y*v.y;
                    float term1 = atomDipoleX*u.x*v.x + atomQuadrupoleXY*u.y*v.x + atomQuadrupoleXZ*u.x*v.y;
                    float term2 = atomQuadrupoleXX * u.x * v.x;
                    result += term0*t.x + term1*t.y + term2*t.z;
                }
                if (z1 > gridPoint.z)
                {
                    gridIndex1 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z;
                    gridIndex2 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+gridPoint.z;
                    firstAtom = cSim.pPmeAtomRange[gridIndex1];
                    lastAtom = cSim.pPmeAtomRange[gridIndex2+1];
                    for (int i = firstAtom; i < lastAtom; ++i)
                    {
                        int2 atomData = cSim.pPmeAtomGridIndex[i];
                        int atomIndex = atomData.x;
                        int z = atomData.y;
                        int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : cSim.pmeGridSize.z);
                        if( iz >= cSim.pmeGridSize.z ){
                            iz -= cSim.pmeGridSize.z;
                        }
                        float atomCharge = cSim.pPosq[atomIndex].w;
                        float atomDipoleX = xscale*cAmoebaSim.pLabFrameDipole[atomIndex*3];
                        float atomDipoleY = yscale*cAmoebaSim.pLabFrameDipole[atomIndex*3+1];
                        float atomDipoleZ = zscale*cAmoebaSim.pLabFrameDipole[atomIndex*3+2];
                        float atomQuadrupoleXX = xscale*xscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9];
                        float atomQuadrupoleXY = 2*xscale*yscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+1];
                        float atomQuadrupoleXZ = 2*xscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+2];
                        float atomQuadrupoleYY = yscale*yscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+4];
                        float atomQuadrupoleYZ = 2*yscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+5];
                        float atomQuadrupoleZZ = zscale*zscale*cAmoebaSim.pLabFrameQuadrupole[atomIndex*9+8];
                        float4 t = cAmoebaSim.pThetai1[atomIndex*AMOEBA_PME_ORDER+ix];
                        float4 u = cAmoebaSim.pThetai2[atomIndex*AMOEBA_PME_ORDER+iy];
                        float4 v = cAmoebaSim.pThetai3[atomIndex*AMOEBA_PME_ORDER+iz];
                        float term0 = atomCharge*u.x*v.x + atomDipoleY*u.y*v.x + atomDipoleZ*u.x*v.y + atomQuadrupoleYY*u.z*v.x + atomQuadrupoleZZ*u.x*v.z + atomQuadrupoleYZ*u.y*v.y;
                        float term1 = atomDipoleX*u.x*v.x + atomQuadrupoleXY*u.y*v.x + atomQuadrupoleXZ*u.x*v.y;
                        float term2 = atomQuadrupoleXX * u.x * v.x;
                        result += term0*t.x + term1*t.y + term2*t.z;
                    }
                }
            }
        }
        cSim.pPmeGrid[gridIndex] = make_hipComplex(result, 0.0f);
    }
}

__global__
__launch_bounds__(64, 10)
void kGridSpreadInducedDipoles_kernel()
{
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    unsigned int numGridPoints = cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
    unsigned int numThreads = gridDim.x*blockDim.x;
    for (int gridIndex = blockIdx.x*blockDim.x+threadIdx.x; gridIndex < numGridPoints; gridIndex += numThreads)
    {
        int3 gridPoint;
        gridPoint.x = gridIndex/(cSim.pmeGridSize.y*cSim.pmeGridSize.z);
        int remainder = gridIndex-gridPoint.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
        gridPoint.y = remainder/cSim.pmeGridSize.z;
        gridPoint.z = remainder-gridPoint.y*cSim.pmeGridSize.z;
        hipfftComplex result = make_hipComplex(0.0f, 0.0f);
        for (int ix = 0; ix < AMOEBA_PME_ORDER; ++ix)
        {
            int x = gridPoint.x-ix+(gridPoint.x >= ix ? 0 : cSim.pmeGridSize.x);
            for (int iy = 0; iy < AMOEBA_PME_ORDER; ++iy)
            {
                int y = gridPoint.y-iy+(gridPoint.y >= iy ? 0 : cSim.pmeGridSize.y);
                int z1 = gridPoint.z-AMOEBA_PME_ORDER+1;
                z1 += (z1 >= 0 ? 0 : cSim.pmeGridSize.z);
                int z2 = (z1 < gridPoint.z ? gridPoint.z : cSim.pmeGridSize.z-1);
                int gridIndex1 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+z1;
                int gridIndex2 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+z2;
                int firstAtom = cSim.pPmeAtomRange[gridIndex1];
                int lastAtom = cSim.pPmeAtomRange[gridIndex2+1];
                for (int i = firstAtom; i < lastAtom; ++i)
                {
                    int2 atomData = cSim.pPmeAtomGridIndex[i];
                    int atomIndex = atomData.x;
                    int z = atomData.y;
                    int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : cSim.pmeGridSize.z);
                    if( iz >= cSim.pmeGridSize.z ){
                        iz -= cSim.pmeGridSize.z;
                    }
                    float inducedDipoleX = xscale*cAmoebaSim.pInducedDipole[atomIndex*3];
                    float inducedDipoleY = yscale*cAmoebaSim.pInducedDipole[atomIndex*3+1];
                    float inducedDipoleZ = zscale*cAmoebaSim.pInducedDipole[atomIndex*3+2];
                    float inducedDipolePolarX = xscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3];
                    float inducedDipolePolarY = yscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3+1];
                    float inducedDipolePolarZ = zscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3+2];
                    float4 t = cAmoebaSim.pThetai1[atomIndex*AMOEBA_PME_ORDER+ix];
                    float4 u = cAmoebaSim.pThetai2[atomIndex*AMOEBA_PME_ORDER+iy];
                    float4 v = cAmoebaSim.pThetai3[atomIndex*AMOEBA_PME_ORDER+iz];
                    float term01 = inducedDipoleY*u.y*v.x + inducedDipoleZ*u.x*v.y;
                    float term11 = inducedDipoleX*u.x*v.x;
                    float term02 = inducedDipolePolarY*u.y*v.x + inducedDipolePolarZ*u.x*v.y;
                    float term12 = inducedDipolePolarX*u.x*v.x;
                    result.x += term01*t.x + term11*t.y;
                    result.y += term02*t.x + term12*t.y;
                }
                if (z1 > gridPoint.z)
                {
                    gridIndex1 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z;
                    gridIndex2 = x*cSim.pmeGridSize.y*cSim.pmeGridSize.z+y*cSim.pmeGridSize.z+gridPoint.z;
                    firstAtom = cSim.pPmeAtomRange[gridIndex1];
                    lastAtom = cSim.pPmeAtomRange[gridIndex2+1];
                    for (int i = firstAtom; i < lastAtom; ++i)
                    {
                        int2 atomData = cSim.pPmeAtomGridIndex[i];
                        int atomIndex = atomData.x;
                        int z = atomData.y;
                        int iz = gridPoint.z-z+(gridPoint.z >= z ? 0 : cSim.pmeGridSize.z);
                        if( iz >= cSim.pmeGridSize.z ){
                            iz -= cSim.pmeGridSize.z;
                        }
                        float inducedDipoleX = xscale*cAmoebaSim.pInducedDipole[atomIndex*3];
                        float inducedDipoleY = yscale*cAmoebaSim.pInducedDipole[atomIndex*3+1];
                        float inducedDipoleZ = zscale*cAmoebaSim.pInducedDipole[atomIndex*3+2];
                        float inducedDipolePolarX = xscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3];
                        float inducedDipolePolarY = yscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3+1];
                        float inducedDipolePolarZ = zscale*cAmoebaSim.pInducedDipolePolar[atomIndex*3+2];
                        float4 t = cAmoebaSim.pThetai1[atomIndex*AMOEBA_PME_ORDER+ix];
                        float4 u = cAmoebaSim.pThetai2[atomIndex*AMOEBA_PME_ORDER+iy];
                        float4 v = cAmoebaSim.pThetai3[atomIndex*AMOEBA_PME_ORDER+iz];
                        float term01 = inducedDipoleY*u.y*v.x + inducedDipoleZ*u.x*v.y;
                        float term11 = inducedDipoleX*u.x*v.x;
                        float term02 = inducedDipolePolarY*u.y*v.x + inducedDipolePolarZ*u.x*v.y;
                        float term12 = inducedDipolePolarX*u.x*v.x;
                        result.x += term01*t.x + term11*t.y;
                        result.y += term02*t.x + term12*t.y;
                    }
                }
            }
        }
        cSim.pPmeGrid[gridIndex] = result;
    }
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(768, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(384, 1)
#else
__launch_bounds__(192, 1)
#endif
void kAmoebaReciprocalConvolution_kernel()
{
    const unsigned int gridSize = cSim.pmeGridSize.x*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
    float expFactor = LOCAL_HACK_PI*LOCAL_HACK_PI/(cSim.alphaEwald*cSim.alphaEwald);
    float scaleFactor = 1.0/(LOCAL_HACK_PI*cSim.periodicBoxSizeX*cSim.periodicBoxSizeY*cSim.periodicBoxSizeZ);
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x)
    {
        int kx = index/(cSim.pmeGridSize.y*cSim.pmeGridSize.z);
        int remainder = index-kx*cSim.pmeGridSize.y*cSim.pmeGridSize.z;
        int ky = remainder/cSim.pmeGridSize.z;
        int kz = remainder-ky*cSim.pmeGridSize.z;
        if (kx == 0 && ky == 0 && kz == 0){
            cSim.pPmeGrid[index] = make_hipComplex(0.0f, 0.0f);
            continue;
        }
        int mx = (kx < (cSim.pmeGridSize.x+1)/2) ? kx : (kx-cSim.pmeGridSize.x);
        int my = (ky < (cSim.pmeGridSize.y+1)/2) ? ky : (ky-cSim.pmeGridSize.y);
        int mz = (kz < (cSim.pmeGridSize.z+1)/2) ? kz : (kz-cSim.pmeGridSize.z);
        float mhx = mx*cSim.invPeriodicBoxSizeX;
        float mhy = my*cSim.invPeriodicBoxSizeY;
        float mhz = mz*cSim.invPeriodicBoxSizeZ;
        float bx = cSim.pPmeBsplineModuli[0][kx];
        float by = cSim.pPmeBsplineModuli[1][ky];
        float bz = cSim.pPmeBsplineModuli[2][kz];
        hipComplex grid = cSim.pPmeGrid[index];
        float m2 = mhx*mhx+mhy*mhy+mhz*mhz;
        float denom = m2*bx*by*bz;
        float eterm = scaleFactor*expf(-expFactor*m2)/denom;
        cSim.pPmeGrid[index] = make_hipComplex(grid.x*eterm, grid.y*eterm);
    }
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(384, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(192, 1)
#else
__launch_bounds__(96, 1)
#endif
void kComputeFixedPotentialFromGrid_kernel()
{
    // extract the permanent multipole field at each site

    for (int m = blockIdx.x*blockDim.x+threadIdx.x; m < cSim.atoms; m += blockDim.x*gridDim.x) {
        int4 gridPoint = cAmoebaSim.pIgrid[m];
        float tuv000 = 0.0f;
        float tuv001 = 0.0f;
        float tuv010 = 0.0f;
        float tuv100 = 0.0f;
        float tuv200 = 0.0f;
        float tuv020 = 0.0f;
        float tuv002 = 0.0f;
        float tuv110 = 0.0f;
        float tuv101 = 0.0f;
        float tuv011 = 0.0f;
        float tuv300 = 0.0f;
        float tuv030 = 0.0f;
        float tuv003 = 0.0f;
        float tuv210 = 0.0f;
        float tuv201 = 0.0f;
        float tuv120 = 0.0f;
        float tuv021 = 0.0f;
        float tuv102 = 0.0f;
        float tuv012 = 0.0f;
        float tuv111 = 0.0f;
        for (int iz = 0; iz < AMOEBA_PME_ORDER; iz++) {
            int k = gridPoint.z+iz-(gridPoint.z+iz >= cSim.pmeGridSize.z ? cSim.pmeGridSize.z : 0);
            float4 v = cAmoebaSim.pThetai3[m*AMOEBA_PME_ORDER+iz];
            float tu00 = 0.0f;
            float tu10 = 0.0f;
            float tu01 = 0.0f;
            float tu20 = 0.0f;
            float tu11 = 0.0f;
            float tu02 = 0.0f;
            float tu30 = 0.0f;
            float tu21 = 0.0f;
            float tu12 = 0.0f;
            float tu03 = 0.0f;
            for (int iy = 0; iy < AMOEBA_PME_ORDER; iy++) {
                int j = gridPoint.y+iy-(gridPoint.y+iy >= cSim.pmeGridSize.y ? cSim.pmeGridSize.y : 0);
                float4 u = cAmoebaSim.pThetai2[m*AMOEBA_PME_ORDER+iy];
                float4 t = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
                for (int ix = 0; ix < AMOEBA_PME_ORDER; ix++) {
                    int i = gridPoint.x+ix-(gridPoint.x+ix >= cSim.pmeGridSize.x ? cSim.pmeGridSize.x : 0);
                    int gridIndex = i*cSim.pmeGridSize.y*cSim.pmeGridSize.z + j*cSim.pmeGridSize.z + k;
                    float tq = cSim.pPmeGrid[gridIndex].x;
                    float4 tadd = cAmoebaSim.pThetai1[m*AMOEBA_PME_ORDER+ix];
                    t.x += tq*tadd.x;
                    t.y += tq*tadd.y;
                    t.z += tq*tadd.z;
                    t.w += tq*tadd.w;
                }
                tu00 += t.x*u.x;
                tu10 += t.y*u.x;
                tu01 += t.x*u.y;
                tu20 += t.z*u.x;
                tu11 += t.y*u.y;
                tu02 += t.x*u.z;
                tu30 += t.w*u.x;
                tu21 += t.z*u.y;
                tu12 += t.y*u.z;
                tu03 += t.x*u.w;
            }
            tuv000 += tu00*v.x;
            tuv100 += tu10*v.x;
            tuv010 += tu01*v.x;
            tuv001 += tu00*v.y;
            tuv200 += tu20*v.x;
            tuv020 += tu02*v.x;
            tuv002 += tu00*v.z;
            tuv110 += tu11*v.x;
            tuv101 += tu10*v.y;
            tuv011 += tu01*v.y;
            tuv300 += tu30*v.x;
            tuv030 += tu03*v.x;
            tuv003 += tu00*v.w;
            tuv210 += tu21*v.x;
            tuv201 += tu20*v.y;
            tuv120 += tu12*v.x;
            tuv021 += tu02*v.y;
            tuv102 += tu10*v.z;
            tuv012 += tu01*v.z;
            tuv111 += tu11*v.y;
        }
        cAmoebaSim.pPhi[20*m] = tuv000;
        cAmoebaSim.pPhi[20*m+1] = tuv100;
        cAmoebaSim.pPhi[20*m+2] = tuv010;
        cAmoebaSim.pPhi[20*m+3] = tuv001;
        cAmoebaSim.pPhi[20*m+4] = tuv200;
        cAmoebaSim.pPhi[20*m+5] = tuv020;
        cAmoebaSim.pPhi[20*m+6] = tuv002;
        cAmoebaSim.pPhi[20*m+7] = tuv110;
        cAmoebaSim.pPhi[20*m+8] = tuv101;
        cAmoebaSim.pPhi[20*m+9] = tuv011;
        cAmoebaSim.pPhi[20*m+10] = tuv300;
        cAmoebaSim.pPhi[20*m+11] = tuv030;
        cAmoebaSim.pPhi[20*m+12] = tuv003;
        cAmoebaSim.pPhi[20*m+13] = tuv210;
        cAmoebaSim.pPhi[20*m+14] = tuv201;
        cAmoebaSim.pPhi[20*m+15] = tuv120;
        cAmoebaSim.pPhi[20*m+16] = tuv021;
        cAmoebaSim.pPhi[20*m+17] = tuv102;
        cAmoebaSim.pPhi[20*m+18] = tuv012;
        cAmoebaSim.pPhi[20*m+19] = tuv111;
    }
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(256, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(128, 1)
#else
__launch_bounds__(64, 1)
#endif
void kComputeInducedPotentialFromGrid_kernel()
{
    // extract the induced dipole field at each site

    for (int m = blockIdx.x*blockDim.x+threadIdx.x; m < cSim.atoms; m += blockDim.x*gridDim.x) {
        int4 gridPoint = cAmoebaSim.pIgrid[m];
        float tuv100_1 = 0.0f;
        float tuv010_1 = 0.0f;
        float tuv001_1 = 0.0f;
        float tuv200_1 = 0.0f;
        float tuv020_1 = 0.0f;
        float tuv002_1 = 0.0f;
        float tuv110_1 = 0.0f;
        float tuv101_1 = 0.0f;
        float tuv011_1 = 0.0f;
        float tuv100_2 = 0.0f;
        float tuv010_2 = 0.0f;
        float tuv001_2 = 0.0f;
        float tuv200_2 = 0.0f;
        float tuv020_2 = 0.0f;
        float tuv002_2 = 0.0f;
        float tuv110_2 = 0.0f;
        float tuv101_2 = 0.0f;
        float tuv011_2 = 0.0f;
        float tuv000 = 0.0f;
        float tuv001 = 0.0f;
        float tuv010 = 0.0f;
        float tuv100 = 0.0f;
        float tuv200 = 0.0f;
        float tuv020 = 0.0f;
        float tuv002 = 0.0f;
        float tuv110 = 0.0f;
        float tuv101 = 0.0f;
        float tuv011 = 0.0f;
        float tuv300 = 0.0f;
        float tuv030 = 0.0f;
        float tuv003 = 0.0f;
        float tuv210 = 0.0f;
        float tuv201 = 0.0f;
        float tuv120 = 0.0f;
        float tuv021 = 0.0f;
        float tuv102 = 0.0f;
        float tuv012 = 0.0f;
        float tuv111 = 0.0f;
        for (int iz = 0; iz < AMOEBA_PME_ORDER; iz++) {
            int k = gridPoint.z+iz-(gridPoint.z+iz >= cSim.pmeGridSize.z ? cSim.pmeGridSize.z : 0);
            float4 v = cAmoebaSim.pThetai3[m*AMOEBA_PME_ORDER+iz];
            float tu00_1 = 0.0f;
            float tu01_1 = 0.0f;
            float tu10_1 = 0.0f;
            float tu20_1 = 0.0f;
            float tu11_1 = 0.0f;
            float tu02_1 = 0.0f;
            float tu00_2 = 0.0f;
            float tu01_2 = 0.0f;
            float tu10_2 = 0.0f;
            float tu20_2 = 0.0f;
            float tu11_2 = 0.0f;
            float tu02_2 = 0.0f;
            float tu00 = 0.0f;
            float tu10 = 0.0f;
            float tu01 = 0.0f;
            float tu20 = 0.0f;
            float tu11 = 0.0f;
            float tu02 = 0.0f;
            float tu30 = 0.0f;
            float tu21 = 0.0f;
            float tu12 = 0.0f;
            float tu03 = 0.0f;
            for (int iy = 0; iy < AMOEBA_PME_ORDER; iy++) {
                int j = gridPoint.y+iy-(gridPoint.y+iy >= cSim.pmeGridSize.y ? cSim.pmeGridSize.y : 0);
                float4 u = cAmoebaSim.pThetai2[m*AMOEBA_PME_ORDER+iy];
                float t0_1 = 0.0f;
                float t1_1 = 0.0f;
                float t2_1 = 0.0f;
                float t0_2 = 0.0f;
                float t1_2 = 0.0f;
                float t2_2 = 0.0f;
                float t3 = 0.0f;
                for (int ix = 0; ix < AMOEBA_PME_ORDER; ix++) {
                    int i = gridPoint.x+ix-(gridPoint.x+ix >= cSim.pmeGridSize.x ? cSim.pmeGridSize.x : 0);
                    int gridIndex = i*cSim.pmeGridSize.y*cSim.pmeGridSize.z + j*cSim.pmeGridSize.z + k;
                    hipfftComplex tq = cSim.pPmeGrid[gridIndex];
                    float4 tadd = cAmoebaSim.pThetai1[m*AMOEBA_PME_ORDER+ix];
                    t0_1 += tq.x*tadd.x;
                    t1_1 += tq.x*tadd.y;
                    t2_1 += tq.x*tadd.z;
                    t0_2 += tq.y*tadd.x;
                    t1_2 += tq.y*tadd.y;
                    t2_2 += tq.y*tadd.z;
                    t3 += (tq.x+tq.y)*tadd.w;
                }
                tu00_1 += t0_1*u.x;
                tu10_1 += t1_1*u.x;
                tu01_1 += t0_1*u.y;
                tu20_1 += t2_1*u.x;
                tu11_1 += t1_1*u.y;
                tu02_1 += t0_1*u.z;
                tu00_2 += t0_2*u.x;
                tu10_2 += t1_2*u.x;
                tu01_2 += t0_2*u.y;
                tu20_2 += t2_2*u.x;
                tu11_2 += t1_2*u.y;
                tu02_2 += t0_2*u.z;
                float t0 = t0_1 + t0_2;
                float t1 = t1_1 + t1_2;
                float t2 = t2_1 + t2_2;
                tu00 += t0*u.x;
                tu10 += t1*u.x;
                tu01 += t0*u.y;
                tu20 += t2*u.x;
                tu11 += t1*u.y;
                tu02 += t0*u.z;
                tu30 += t3*u.x;
                tu21 += t2*u.y;
                tu12 += t1*u.z;
                tu03 += t0*u.w;
            }
            tuv100_1 += tu10_1*v.x;
            tuv010_1 += tu01_1*v.x;
            tuv001_1 += tu00_1*v.y;
            tuv200_1 += tu20_1*v.x;
            tuv020_1 += tu02_1*v.x;
            tuv002_1 += tu00_1*v.z;
            tuv110_1 += tu11_1*v.x;
            tuv101_1 += tu10_1*v.y;
            tuv011_1 += tu01_1*v.y;
            tuv100_2 += tu10_2*v.x;
            tuv010_2 += tu01_2*v.x;
            tuv001_2 += tu00_2*v.y;
            tuv200_2 += tu20_2*v.x;
            tuv020_2 += tu02_2*v.x;
            tuv002_2 += tu00_2*v.z;
            tuv110_2 += tu11_2*v.x;
            tuv101_2 += tu10_2*v.y;
            tuv011_2 += tu01_2*v.y;
            tuv000 += tu00*v.x;
            tuv100 += tu10*v.x;
            tuv010 += tu01*v.x;
            tuv001 += tu00*v.y;
            tuv200 += tu20*v.x;
            tuv020 += tu02*v.x;
            tuv002 += tu00*v.z;
            tuv110 += tu11*v.x;
            tuv101 += tu10*v.y;
            tuv011 += tu01*v.y;
            tuv300 += tu30*v.x;
            tuv030 += tu03*v.x;
            tuv003 += tu00*v.w;
            tuv210 += tu21*v.x;
            tuv201 += tu20*v.y;
            tuv120 += tu12*v.x;
            tuv021 += tu02*v.y;
            tuv102 += tu10*v.z;
            tuv012 += tu01*v.z;
            tuv111 += tu11*v.y;
        }
        cAmoebaSim.pPhid[10*m]   = 0.0f;
        cAmoebaSim.pPhid[10*m+1] = tuv100_1;
        cAmoebaSim.pPhid[10*m+2] = tuv010_1;
        cAmoebaSim.pPhid[10*m+3] = tuv001_1;
        cAmoebaSim.pPhid[10*m+4] = tuv200_1;
        cAmoebaSim.pPhid[10*m+5] = tuv020_1;
        cAmoebaSim.pPhid[10*m+6] = tuv002_1;
        cAmoebaSim.pPhid[10*m+7] = tuv110_1;
        cAmoebaSim.pPhid[10*m+8] = tuv101_1;
        cAmoebaSim.pPhid[10*m+9] = tuv011_1;

        cAmoebaSim.pPhip[10*m]   = 0.0f;
        cAmoebaSim.pPhip[10*m+1] = tuv100_2;
        cAmoebaSim.pPhip[10*m+2] = tuv010_2;
        cAmoebaSim.pPhip[10*m+3] = tuv001_2;
        cAmoebaSim.pPhip[10*m+4] = tuv200_2;
        cAmoebaSim.pPhip[10*m+5] = tuv020_2;
        cAmoebaSim.pPhip[10*m+6] = tuv002_2;
        cAmoebaSim.pPhip[10*m+7] = tuv110_2;
        cAmoebaSim.pPhip[10*m+8] = tuv101_2;
        cAmoebaSim.pPhip[10*m+9] = tuv011_2;

        cAmoebaSim.pPhidp[20*m] = tuv000;
        cAmoebaSim.pPhidp[20*m+1] = tuv100;
        cAmoebaSim.pPhidp[20*m+2] = tuv010;
        cAmoebaSim.pPhidp[20*m+3] = tuv001;
        cAmoebaSim.pPhidp[20*m+4] = tuv200;
        cAmoebaSim.pPhidp[20*m+5] = tuv020;
        cAmoebaSim.pPhidp[20*m+6] = tuv002;
        cAmoebaSim.pPhidp[20*m+7] = tuv110;
        cAmoebaSim.pPhidp[20*m+8] = tuv101;
        cAmoebaSim.pPhidp[20*m+9] = tuv011;
        cAmoebaSim.pPhidp[20*m+10] = tuv300;
        cAmoebaSim.pPhidp[20*m+11] = tuv030;
        cAmoebaSim.pPhidp[20*m+12] = tuv003;
        cAmoebaSim.pPhidp[20*m+13] = tuv210;
        cAmoebaSim.pPhidp[20*m+14] = tuv201;
        cAmoebaSim.pPhidp[20*m+15] = tuv120;
        cAmoebaSim.pPhidp[20*m+16] = tuv021;
        cAmoebaSim.pPhidp[20*m+17] = tuv102;
        cAmoebaSim.pPhidp[20*m+18] = tuv012;
        cAmoebaSim.pPhidp[20*m+19] = tuv111;
    }
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(768, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(384, 1)
#else
__launch_bounds__(192, 1)
#endif
void kComputeFixedMultipoleForceAndEnergy_kernel()
{
    float multipole[10];
    const int deriv1[] = {1, 4, 7, 8, 10, 15, 17, 13, 14, 19};
    const int deriv2[] = {2, 7, 5, 9, 13, 11, 18, 15, 19, 16};
    const int deriv3[] = {3, 8, 9, 6, 14, 16, 12, 19, 17, 18};
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    float energy = 0.0f;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < cSim.atoms; i += blockDim.x*gridDim.x) {
        // Compute the torque.

        multipole[0] = cSim.pPosq[i].w;
        multipole[1] = cAmoebaSim.pLabFrameDipole[i*3];
        multipole[2] = cAmoebaSim.pLabFrameDipole[i*3+1];
        multipole[3] = cAmoebaSim.pLabFrameDipole[i*3+2];
        multipole[4] = cAmoebaSim.pLabFrameQuadrupole[i*9];
        multipole[5] = cAmoebaSim.pLabFrameQuadrupole[i*9+4];
        multipole[6] = cAmoebaSim.pLabFrameQuadrupole[i*9+8];
        multipole[7] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+1];
        multipole[8] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+2];
        multipole[9] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+5];

        float* phi = &cAmoebaSim.pPhi[20*i];

        cAmoebaSim.pTorque[3*i] = cAmoebaSim.electric*(multipole[3]*yscale*phi[2] - multipole[2]*zscale*phi[3]
                      + 2.0f*(multipole[6]-multipole[5])*yscale*zscale*phi[9]
                      + multipole[8]*xscale*yscale*phi[7] + multipole[9]*yscale*yscale*phi[5]
                      - multipole[7]*xscale*zscale*phi[8] - multipole[9]*zscale*zscale*phi[6]);

        cAmoebaSim.pTorque[3*i+1] = cAmoebaSim.electric*(multipole[1]*zscale*phi[3] - multipole[3]*xscale*phi[1]
                      + 2.0f*(multipole[4]-multipole[6])*xscale*zscale*phi[8]
                      + multipole[7]*yscale*zscale*phi[9] + multipole[8]*zscale*zscale*phi[6]
                      - multipole[8]*xscale*xscale*phi[4] - multipole[9]*xscale*yscale*phi[7]);

        cAmoebaSim.pTorque[3*i+2] = cAmoebaSim.electric*(multipole[2]*xscale*phi[1] - multipole[1]*yscale*phi[2]
                      + 2.0f*(multipole[5]-multipole[4])*xscale*yscale*phi[7]
                      + multipole[7]*xscale*xscale*phi[4] + multipole[9]*xscale*zscale*phi[8]
                      - multipole[7]*yscale*yscale*phi[5] - multipole[8]*yscale*zscale*phi[9]);

        // Compute the force and energy.

        multipole[1] *= xscale;
        multipole[2] *= yscale;
        multipole[3] *= zscale;
        multipole[4] *= xscale*xscale;
        multipole[5] *= yscale*yscale;
        multipole[6] *= zscale*zscale;
        multipole[7] *= xscale*yscale;
        multipole[8] *= xscale*zscale;
        multipole[9] *= yscale*zscale;

        float4 f = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        for (int k = 0; k < 10; k++) {
            energy += multipole[k]*phi[k];
            f.x += multipole[k]*phi[deriv1[k]];
            f.y += multipole[k]*phi[deriv2[k]];
            f.z += multipole[k]*phi[deriv3[k]];
        }
        f.x *= cAmoebaSim.electric*xscale;
        f.y *= cAmoebaSim.electric*yscale;
        f.z *= cAmoebaSim.electric*zscale;
        float4 force = cSim.pForce4[i];
        force.x -= f.x;
        force.y -= f.y;
        force.z -= f.z;
        cSim.pForce4[i] = force;

    }
    cSim.pEnergy[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*cAmoebaSim.electric*energy;
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(768, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(384, 1)
#else
__launch_bounds__(192, 1)
#endif
void kComputeInducedDipoleForceAndEnergy_kernel()
{
    float multipole[10];
    float inducedDipole[3];
    float inducedDipolePolar[3];
    float scales[3];
    const int deriv1[] = {1, 4, 7, 8, 10, 15, 17, 13, 14, 19};
    const int deriv2[] = {2, 7, 5, 9, 13, 11, 18, 15, 19, 16};
    const int deriv3[] = {3, 8, 9, 6, 14, 16, 12, 19, 17, 18};
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    scales[0] = xscale;
    scales[1] = yscale;
    scales[2] = zscale;
    float energy = 0.0f;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < cSim.atoms; i += blockDim.x*gridDim.x) {
        // Compute the torque.

        multipole[0] = cSim.pPosq[i].w;
        multipole[1] = cAmoebaSim.pLabFrameDipole[i*3];
        multipole[2] = cAmoebaSim.pLabFrameDipole[i*3+1];
        multipole[3] = cAmoebaSim.pLabFrameDipole[i*3+2];
        multipole[4] = cAmoebaSim.pLabFrameQuadrupole[i*9];
        multipole[5] = cAmoebaSim.pLabFrameQuadrupole[i*9+4];
        multipole[6] = cAmoebaSim.pLabFrameQuadrupole[i*9+8];
        multipole[7] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+1];
        multipole[8] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+2];
        multipole[9] = 2*cAmoebaSim.pLabFrameQuadrupole[i*9+5];
        float* phidp = &cAmoebaSim.pPhidp[20*i];
 
        cAmoebaSim.pTorque[3*i] += 0.5f*cAmoebaSim.electric*(multipole[3]*yscale*phidp[2] - multipole[2]*zscale*phidp[3]
                      + 2.0f*(multipole[6]-multipole[5])*yscale*zscale*phidp[9]
                      + multipole[8]*xscale*yscale*phidp[7] + multipole[9]*yscale*yscale*phidp[5]
                      - multipole[7]*xscale*zscale*phidp[8] - multipole[9]*zscale*zscale*phidp[6]);

        cAmoebaSim.pTorque[3*i+1] += 0.5f*cAmoebaSim.electric*(multipole[1]*zscale*phidp[3] - multipole[3]*xscale*phidp[1]
                      + 2.0f*(multipole[4]-multipole[6])*xscale*zscale*phidp[8]
                      + multipole[7]*yscale*zscale*phidp[9] + multipole[8]*zscale*zscale*phidp[6]
                      - multipole[8]*xscale*xscale*phidp[4] - multipole[9]*xscale*yscale*phidp[7]);

        cAmoebaSim.pTorque[3*i+2] += 0.5f*cAmoebaSim.electric*(multipole[2]*xscale*phidp[1] - multipole[1]*yscale*phidp[2]
                      + 2.0f*(multipole[5]-multipole[4])*xscale*yscale*phidp[7]
                      + multipole[7]*xscale*xscale*phidp[4] + multipole[9]*xscale*zscale*phidp[8]
                      - multipole[7]*yscale*yscale*phidp[5] - multipole[8]*yscale*zscale*phidp[9]);

        // Compute the force and energy.

        multipole[1] *= xscale;
        multipole[2] *= yscale;
        multipole[3] *= zscale;
        multipole[4] *= xscale*xscale;
        multipole[5] *= yscale*yscale;
        multipole[6] *= zscale*zscale;
        multipole[7] *= xscale*yscale;
        multipole[8] *= xscale*zscale;
        multipole[9] *= yscale*zscale;

        inducedDipole[0] = cAmoebaSim.pInducedDipole[i*3];
        inducedDipole[1] = cAmoebaSim.pInducedDipole[i*3+1];
        inducedDipole[2] = cAmoebaSim.pInducedDipole[i*3+2];
        inducedDipolePolar[0] = cAmoebaSim.pInducedDipolePolar[i*3];
        inducedDipolePolar[1] = cAmoebaSim.pInducedDipolePolar[i*3+1];
        inducedDipolePolar[2] = cAmoebaSim.pInducedDipolePolar[i*3+2];
        float* phi = &cAmoebaSim.pPhi[20*i];
        float* phip = &cAmoebaSim.pPhip[10*i];
        float* phid = &cAmoebaSim.pPhid[10*i];
        float4 f = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        energy += cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX*inducedDipole[0]*phi[1];
        energy += cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY*inducedDipole[1]*phi[2];
        energy += cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ*inducedDipole[2]*phi[3];

        for (int k = 0; k < 3; k++) {

            int j1 = deriv1[k+1];
            int j2 = deriv2[k+1];
            int j3 = deriv3[k+1];

            f.x += (inducedDipole[k]+inducedDipolePolar[k])*phi[j1]*(scales[k]/xscale);
            f.y += (inducedDipole[k]+inducedDipolePolar[k])*phi[j2]*(scales[k]/yscale);
            f.z += (inducedDipole[k]+inducedDipolePolar[k])*phi[j3]*(scales[k]/zscale);
 
            if( cAmoebaSim.polarizationType == 0 )
            {
                f.x += (inducedDipole[k]*phip[j1] + inducedDipolePolar[k]*phid[j1])*(scales[k]/xscale);
                f.y += (inducedDipole[k]*phip[j2] + inducedDipolePolar[k]*phid[j2])*(scales[k]/yscale);
                f.z += (inducedDipole[k]*phip[j3] + inducedDipolePolar[k]*phid[j3])*(scales[k]/zscale);
            }


        }

        f.x *= cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
        f.y *= cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
        f.z *= cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
        for (int k = 0; k < 10; k++) {
            f.x += multipole[k]*phidp[deriv1[k]];
            f.y += multipole[k]*phidp[deriv2[k]];
            f.z += multipole[k]*phidp[deriv3[k]];
        }

        f.x *= 0.5f*cAmoebaSim.electric*xscale;
        f.y *= 0.5f*cAmoebaSim.electric*yscale;
        f.z *= 0.5f*cAmoebaSim.electric*zscale;

        float4 force = cSim.pForce4[i];
        force.x -= f.x;
        force.y -= f.y;
        force.z -= f.z;
        cSim.pForce4[i] = force;
    }
    cSim.pEnergy[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*cAmoebaSim.electric*energy;
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(768, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(384, 1)
#else
__launch_bounds__(192, 1)
#endif
void kRecordFixedMultipoleField_kernel(float* output)
{
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < cSim.atoms; i += blockDim.x*gridDim.x) {
        output[3*i] = -xscale*cAmoebaSim.pPhi[20*i+1];
        output[3*i+1] = -yscale*cAmoebaSim.pPhi[20*i+2];
        output[3*i+2] = -zscale*cAmoebaSim.pPhi[20*i+3];
    }
}

__global__
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(768, 1)
#elif (__CUDA_ARCH__ >= 120)
__launch_bounds__(384, 1)
#else
__launch_bounds__(192, 1)
#endif
void kRecordInducedDipoleField_kernel(float* output, float* outputPolar)
{
    const float xscale = cSim.pmeGridSize.x*cSim.invPeriodicBoxSizeX;
    const float yscale = cSim.pmeGridSize.y*cSim.invPeriodicBoxSizeY;
    const float zscale = cSim.pmeGridSize.z*cSim.invPeriodicBoxSizeZ;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < cSim.atoms; i += blockDim.x*gridDim.x) {
        output[3*i] -= xscale*cAmoebaSim.pPhid[10*i+1];
        output[3*i+1] -= yscale*cAmoebaSim.pPhid[10*i+2];
        output[3*i+2] -= zscale*cAmoebaSim.pPhid[10*i+3];
        outputPolar[3*i] -= xscale*cAmoebaSim.pPhip[10*i+1];
        outputPolar[3*i+1] -= yscale*cAmoebaSim.pPhip[10*i+2];
        outputPolar[3*i+2] -= zscale*cAmoebaSim.pPhip[10*i+3];
    }
}

extern void cudaComputeAmoebaMapTorqueAndAddToForce(amoebaGpuContext gpu, CUDAStream<float>* psTorque);

/**
 * Compute the potential and forces due to the reciprocal space PME calculation for fixed multipoles.
 */
void kCalculateAmoebaPMEFixedMultipoles(amoebaGpuContext amoebaGpu)
{
    // Compute B-spline coefficients and sort the atoms.

    gpuContext gpu = amoebaGpu->gpuContext;
    int bsplineThreads = (gpu->sm_version >= SM_20 ? 448 : (gpu->sm_version >= SM_12 ? 160 : 160));
    kComputeAmoebaBsplines_kernel<<<gpu->sim.blocks, bsplineThreads, bsplineThreads*AMOEBA_PME_ORDER*AMOEBA_PME_ORDER*sizeof(float)>>>();
    LAUNCHERROR("kComputeAmoebaBsplines");
    bbSort(gpu->psPmeAtomGridIndex->_pDevData, gpu->natoms);
    kFindAmoebaAtomRangeForGrid_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kFindAmoebaAtomRangeForGrid");

    // Perform PME for the fixed multipoles.

    kGridSpreadFixedMultipoles_kernel<<<10*gpu->sim.blocks, 64>>>();
    LAUNCHERROR("kGridSpreadFixedMultipoles");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_FORWARD);
    kAmoebaReciprocalConvolution_kernel<<<gpu->sim.blocks, gpu->sim.nonbond_threads_per_block>>>();
    LAUNCHERROR("kAmoebaReciprocalConvolution");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_BACKWARD);
    int potentialThreads = (gpu->sm_version >= SM_20 ? 384 : (gpu->sm_version >= SM_12 ? 192 : 96));
    kComputeFixedPotentialFromGrid_kernel<<<gpu->sim.blocks, potentialThreads>>>();
    LAUNCHERROR("kComputeFixedPotentialFromGrid");
    kRecordFixedMultipoleField_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>(amoebaGpu->psE_Field->_pDevData);
    LAUNCHERROR("kRecordFixedMultipoleField");
    kComputeFixedMultipoleForceAndEnergy_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kComputeFixedMultipoleForceAndEnergy");

}

/**
 * Compute the potential due to the reciprocal space PME calculation for induced dipoles.
 */
void kCalculateAmoebaPMEInducedDipoleField(amoebaGpuContext amoebaGpu)
{
    // Perform PME for the induced dipoles.

    gpuContext gpu = amoebaGpu->gpuContext;
    kGridSpreadInducedDipoles_kernel<<<10*gpu->sim.blocks, 64>>>();
    LAUNCHERROR("kGridSpreadInducedDipoles");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_FORWARD);
    kAmoebaReciprocalConvolution_kernel<<<gpu->sim.blocks, gpu->sim.nonbond_threads_per_block>>>();
    LAUNCHERROR("kAmoebaReciprocalConvolution");
    hipfftExecC2C(gpu->fftplan, gpu->psPmeGrid->_pDevData, gpu->psPmeGrid->_pDevData, HIPFFT_BACKWARD);
    int potentialThreads = (gpu->sm_version >= SM_20 ? 256 : (gpu->sm_version >= SM_12 ? 128 : 64));
    kComputeInducedPotentialFromGrid_kernel<<<gpu->sim.blocks, potentialThreads>>>();
    LAUNCHERROR("kComputeInducedPotentialFromGrid");
    kRecordInducedDipoleField_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>(amoebaGpu->psWorkVector[0]->_pDevData, amoebaGpu->psWorkVector[1]->_pDevData);
    LAUNCHERROR("kRecordInducedDipoleField");
}

/**
 * Compute the forces due to the reciprocal space PME calculation for induced dipoles.
 */
void kCalculateAmoebaPMEInducedDipoleForces(amoebaGpuContext amoebaGpu)
{
    gpuContext gpu = amoebaGpu->gpuContext;
    kComputeInducedDipoleForceAndEnergy_kernel<<<gpu->sim.blocks, gpu->sim.update_threads_per_block>>>();
    LAUNCHERROR("kComputeInducedDipoleForceAndEnergy");

}
