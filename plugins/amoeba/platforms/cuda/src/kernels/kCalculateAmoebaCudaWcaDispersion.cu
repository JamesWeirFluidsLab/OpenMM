#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "amoebaGpuTypes.h"
#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"
#include "kCalculateAmoebaCudaWcaDispersionParticle.h"
#include "amoebaScaleFactors.h"

#include <stdio.h>

using namespace std;

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaCudaWcaDispersionSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaWcaDispersionSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaWcaDispersionFieldSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaCudaWcaDispersionSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaWcaDispersionSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaWcaDispersionSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

__device__ void zeroWcaDispersionSharedForce( struct WcaDispersionParticle* sA ) 
{
    // zero shared fields

    sA->force[0]              = 0.0f;
    sA->force[1]              = 0.0f;
    sA->force[2]              = 0.0f;

}

__device__ void loadWcaDispersionShared( struct WcaDispersionParticle* sA, unsigned int atomI,
                                         float4* atomCoord, float2* wcaParameters )
{
    // coordinates, radius, epsilon

    sA->x                        = atomCoord[atomI].x;
    sA->y                        = atomCoord[atomI].y;
    sA->z                        = atomCoord[atomI].z;

    sA->radius                   = wcaParameters[atomI].x;
    sA->epsilon                  = wcaParameters[atomI].y;

}

// load struct and arrays w/ shared data in sA

__device__ void loadWcaDispersionData( struct WcaDispersionParticle* sA,
                                       float4* jCoord, float* jRadius, float* jEpsilon )
{

    // load coordinates, radius, epsilon

    jCoord->x               = sA->x;
    jCoord->y               = sA->y;
    jCoord->z               = sA->z;

    *jRadius                = sA->radius;
    *jEpsilon               = sA->epsilon;
}

__device__ void calculateWcaDispersionInit_kernel( float iRadius,   float iEpsilon,
                                                   float* rmixo,    float* rmixh,
                                                   float* emixo,    float* emixh

 )
{

    float sqrtEps                   = sqrtf(iEpsilon);
    float denominator               = sqrtf(cAmoebaSim.epso) + sqrtEps;
         *emixo                     = 4.0f*cAmoebaSim.epso*iEpsilon / (denominator*denominator);

          denominator               = sqrtf(cAmoebaSim.epsh) + sqrtEps;
         *emixh                     = 4.0f*cAmoebaSim.epsh*iEpsilon / (denominator*denominator);

    float iRadius2                  = iRadius*iRadius;
    float rmino2                    = cAmoebaSim.rmino*cAmoebaSim.rmino; 
         *rmixo                     = 2.0f*(rmino2*cAmoebaSim.rmino + iRadius2*iRadius) / (rmino2 + iRadius2);

    float rminh2                    = cAmoebaSim.rminh*cAmoebaSim.rminh;
         *rmixh                     = 2.0f*(rminh2*cAmoebaSim.rminh + iRadius2*iRadius) / (rminh2+iRadius2);

}

__device__ void calculateWcaDispersionPairIxn_kernel( float4 atomCoordinatesI, float4 atomCoordinatesJ,
                                                      float radiusI,  float radiusJ,
                                                      float rmixo,    float rmixh,
                                                      float emixo,    float emixh,
                                                      float force[3], float* energy ) {

    const float pi         = 3.1415926535897f;
    const float shctd      = cAmoebaSim.shctd;
    const float awater     = cAmoebaSim.awater;

    // ---------------------------------------------------------------------------------------
    
    // get deltaR, and r between 2 atoms
    
    force[0]                                     = atomCoordinatesJ.x - atomCoordinatesI.x;
    force[1]                                     = atomCoordinatesJ.y - atomCoordinatesI.y;
    force[2]                                     = atomCoordinatesJ.z - atomCoordinatesI.z;

    float r2                                     = force[0]*force[0] + force[1]*force[1] + force[2]*force[2];
    if( r2 <= 0.0f ){
        force[0] = force[1] = force[2] = *energy = 0.0f;
        return;
    }
    float rI                                     = rsqrtf( r2 );
    float r                                      = 1.0f/rI;

    float sk                                     = radiusJ*shctd;
    float sk2                                    = sk*sk;
    if( radiusI >= (r+sk) ){
        force[0] = force[1] = force[2] = *energy = 0.0f;
        return;
    }

    float rmax                                   = radiusI > (r - sk) ? radiusI : (r - sk);
    float lik                                    = rmax;
    float lik2                                   = lik*lik;
    float lik3                                   = lik2*lik;
    float lik4                                   = lik2*lik2;
 
    float uik                                    = (r+sk) < rmixo ? (r+sk) : rmixo;
    float uik2                                   = uik*uik;
    float uik3                                   = uik2*uik;
    float uik4                                   = uik2*uik2;

    // 3453
    float term                                   = 4.0f*pi/(48.f*r)*(3.0f*(lik4-uik4) - 8.0f*r*(lik3-uik3) + 6.0f*(r2-sk2)*(lik2-uik2));

    float r3                                     = r2*r;
    float dl1                                    = lik2*(-lik2 + 2.0f*(r2 + sk2) );
    float dl2                                    = lik*(-lik3 + 4.0f*lik2*r - 6.0f*lik*r2 + 2.0f*lik*sk2 + 4.0f*r3 - 4.0f*r*sk2);
    float dl                                     = radiusI > (r-sk)? dl1 : dl2;

    // 3464

    float du1                                    = uik2*(-uik2 + 2.0f*(r2 + sk2) );
    float du2                                    = uik*(-uik3 + 4.0f*uik2*r - 2.0f*uik*(3.0f*r2 - sk2) + 4.0f*r*(r2 - sk2) );
    //float du2                                    = uik*(uik*( -uik2 + 4.0f*uik*r - 2.0f*(3.0f*r2 - sk2)) + 4.0f*r*(r2 - sk2) );
    float du                                     = (r+sk) > rmixo ? du1 : du2;
          du                                    *= -1.0f;

    float mask2                                  = lik < rmixo ? 1.0f : 0.0f;
    float sum                                    = -mask2*(emixo*term);
    float de                                     = -mask2*emixo*pi*(dl+du)/(4.0f*r2);

    // block at 3476

    uik                                          = (r+sk) < rmixh ? (r+sk) : rmixh;
    uik2                                         = uik*uik;
    uik3                                         = uik2*uik;
    uik4                                         = uik2*uik2;

    // 3481

    term                                         = (pi)/ (12.0f*r) * (3.0f*(lik4-uik4) - 8.0f*r*(lik3-uik3) + 6.0f*(r2-sk2)*(lik2-uik2));

    dl1                                          = lik2*(-lik2 + 2.0f*r2 + 2.0f*sk2);
    dl2                                          = lik*(-lik3 + 4.0f*lik2*r - 6.0f*lik*r2 + 2.0f*lik*sk2 + 4.0f*r3 - 4.0f*r*sk2);
    dl                                           = radiusI > (r-sk) ? dl1 : dl2;

    // 3492

    du1                                          = -uik2*(-uik2 + 2.0f*r2 + 2.0f*sk2);
    du2                                          = -uik*(-uik3 + 4.0f*uik2*r - 6.0f*uik*r2 + 2.0f*uik*sk2 + 4.0f*r3 - 4.0f*r*sk2);
    du                                           = (r+sk) > rmixh ? du1 : du2;

    mask2                                        = lik < rmixh ? 1.0f : 0.0f;
    sum                                         -= mask2*(2.0f*emixh*term);
    de                                          -= mask2*(2.0f*emixh*pi*(dl+du)/(4.0f*r2));

    // 3504

    uik                                          = r + sk;
    uik2                                         = uik*uik;
    uik3                                         = uik2*uik;
    uik4                                         = uik2*uik2;
    float uik5                                   = uik4*uik;
    float uik6                                   = uik3*uik3;
    float uik10                                  = uik5*uik5;
    float uik11                                  = uik10*uik;
    float uik12                                  = uik6*uik6;
    float uik13                                  = uik12*uik;

    lik                                          = rmax > rmixo ? rmax : rmixo;
    lik2                                         = lik*lik;
    lik3                                         = lik2*lik;
    lik4                                         = lik2*lik2;
    float lik5                                   = lik4*lik;
    float lik6                                   = lik3*lik3;
    float lik10                                  = lik5*lik5;
    float lik11                                  = lik10*lik;
    float lik12                                  = lik6*lik6;
    float lik13                                  = lik12*lik;

    // 3525

    term                                         = 4.0f*pi/(120.0f*r*lik5*uik5)*(15.0f*uik*lik*r*(uik4-lik4) - 10.0f*uik2*lik2*(uik3-lik3) + 6.0f*(sk2-r2)*(uik5-lik5));
    dl1                                          = (-5.0f*lik2 + 3.0f*r2 + 3.0f*sk2)/lik5;
    dl2                                          = ( 5.0f*lik3 - 33.0f*lik*r2 - 3.0f*lik*sk2 + 15.0f*(lik2*r+r3-r*sk2))/lik6;
    dl                                           = (radiusI > (r-sk)) || (rmax < rmixo) ? -dl1 : dl2;

    du                                           = (-5.0f*uik3 + 33.0f*uik*r2 + 3.0f*uik*sk2 - 15.0f*(uik2*r+r3-r*sk2))/uik6;

    float rmixo7                                 = rmixo*rmixo*rmixo;
          rmixo7                                 = rmixo7*rmixo7*rmixo;
    float ao                                     = emixo*rmixo7;

    // 3540

    float idisp                                  = -2.0f*ao*term;
    mask2                                        = uik > rmixo ? 1.0f : 0.0f;

    // 3541
    de                                          -= mask2*(2.0f*ao*pi*(dl + du)/(15.0f*r2));

    // 3542

    term                                         = 4.0f*pi/(2640.0f*r*lik12*uik12) * (120.0f*uik*lik*r*(uik11-lik11) - 66.0f*uik2*lik2*(uik10-lik10) + 55.0f*(sk2-r2)*(uik12-lik12));

    // 3546

    dl1                                          = (6.0f*lik2 - 5.0f*r2 - 5.0f*sk2)/lik12;
    dl2                                          = (6.0f*lik3 - 125.0f*lik*r2 - 5.0f*lik*sk2 + 60.0f*(lik2*r+r3-r*sk2))/lik13;
    dl                                           = (radiusI > (r-sk)) || (rmax < rmixo) ? dl1 : dl2;

    // 3554

    du                                           = (-6.0f*uik3 + 125.0f*uik*r2 + 5.0f*uik*sk2 - 60.0f*(uik2*r+r3-r*sk2))/uik13;

    de                                          += mask2*(ao*rmixo7*pi*(dl + du)/(60.0f*r2));
    float irep                                   = ao*rmixo7*term;
    sum                                         += mask2*(irep + idisp);

    // 3562

    lik                                          = rmax > rmixh ? rmax : rmixh;
    lik2                                         = lik*lik;
    lik3                                         = lik2*lik;
    lik4                                         = lik2*lik2;
    lik5                                         = lik4*lik;
    lik6                                         = lik3*lik3;
    lik10                                        = lik5*lik5;
    lik11                                        = lik10*lik;
    lik12                                        = lik6*lik6;
    lik13                                        = lik12*lik;

    // 3572

    term                                         = 4.0f * pi / (120.0f*r*lik5*uik5) * (15.0f*uik*lik*r*(uik4-lik4) -
                                                   10.0f*uik2*lik2*(uik3-lik3) + 6.0f*(sk2-r2)*(uik5-lik5));

    dl1                                          = (-5.0f*lik2 + 3.0f*r2 + 3.0f*sk2)/lik5;
    dl2                                          = (5.0f*lik3 - 33.0f*lik*r2 - 3.0f*lik*sk2+ 15.0f*(lik2*r+r3-r*sk2))/lik6;
    dl                                           = (radiusI > (r-sk)) || (rmax < rmixh) ? -dl1 : dl2;

    du                                           = -(5.0f*uik3 - 33.0f*uik*r2 - 3.0f*uik*sk2 + 15.0f*(uik2*r+r3-r*sk2))/uik6;

    float rmixh7                                 = rmixh*rmixh*rmixh;
          rmixh7                                 = rmixh7*rmixh7*rmixh;
    float ah                                     = emixh * rmixh7;

    // 3587
    idisp                                        = -4.0f * ah * term;

    mask2                                        = uik > rmixh ? 1.0f : 0.0f;
    de                                          -= mask2*(4.0f*ah*pi*(dl + du)/(15.0f*r2));

    term                                         = 4.0f * pi / (2640.0f*r*lik12*uik12) * (120.0f*uik*lik*r*(uik11-lik11) -
                                                   66.0f*uik2*lik2*(uik10-lik10) + 55.0f*(sk2-r2)*(uik12-lik12));

    // 3593

    dl1                                          = -(-6.0f*lik2 + 5.0f*r2 + 5.0f*sk2)/lik12;
    dl2                                          =  (6.0f*lik3 - 125.0f*lik*r2 - 5.0f*lik*sk2 + 60.0f*(lik2*r+r3-r*sk2))/lik13;
    dl                                           = ( (radiusI > (r-sk) ) || (rmax < rmixh) ) ? dl1 : dl2;

    // 3603

    du                                           = -(6.0f*uik3 - 125.0f*uik*r2 -5.0f*uik*sk2 + 60.0f*(uik2*r+r3-r*sk2))/uik13;
    irep                                         = 2.0f*ah*rmixh7*term;

    de                                          += mask2*(ah*rmixh7*pi*(dl+du)/(30.0f*r2));
    sum                                         += mask2*(irep + idisp);

    *energy                                      = sum;

    de                                          *= -(awater/r);
    force[0]                                    *= de;
    force[1]                                    *= de;
    force[2]                                    *= de;

}

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaWcaDispersion.h"

#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaWcaDispersion.h"

/**---------------------------------------------------------------------------------------

   Compute WCA dispersion

   @param amoebaGpu        amoebaGpu context

   --------------------------------------------------------------------------------------- */

void kCalculateAmoebaWcaDispersionForces( amoebaGpuContext amoebaGpu )
{
  
   // ---------------------------------------------------------------------------------------

    static int threadsPerBlock    = 0;

   // ---------------------------------------------------------------------------------------

     gpuContext gpu    = amoebaGpu->gpuContext;

    // set threads/block first time through

    if( threadsPerBlock == 0 ){
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 384;
        else if (gpu->sm_version >= SM_12)
            maxThreads = 192;
        else
            maxThreads = 64;
       threadsPerBlock = std::min(getThreadsPerBlock( amoebaGpu, sizeof(WcaDispersionParticle), gpu->sharedMemoryPerBlock ), maxThreads);
    }

    if (gpu->bOutputBufferPerWarp){

        kCalculateAmoebaWcaDispersionN2ByWarp_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(WcaDispersionParticle)*threadsPerBlock>>>(
                                                            gpu->psWorkUnit->_pDevData );

    } else {

        kCalculateAmoebaWcaDispersionN2_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(WcaDispersionParticle)*threadsPerBlock>>>(
                                                            gpu->psWorkUnit->_pDevData );

    }
    LAUNCHERROR("kCalculateAmoebaWcaDispersion");  

   // ---------------------------------------------------------------------------------------
}
