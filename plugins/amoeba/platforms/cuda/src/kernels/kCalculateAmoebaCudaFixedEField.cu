#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- *
 *                                   OpenMM                                   *
 * -------------------------------------------------------------------------- *
 * This is part of the OpenMM molecular simulation toolkit originating from   *
 * Simbios, the NIH National Center for Physics-Based Simulation of           *
 * Biological Structures at Stanford, funded under the NIH Roadmap for        *
 * Medical Research, grant U54 GM072970. See https://simtk.org.               *
 *                                                                            *
 * Portions copyright (c) 2009 Stanford University and the Authors.           *
 * Authors: Scott Le Grand, Peter Eastman                                     *
 * Contributors:                                                              *
 *                                                                            *
 * This program is free software: you can redistribute it and/or modify       *
 * it under the terms of the GNU Lesser General Public License as published   *
 * by the Free Software Foundation, either version 3 of the License, or       *
 * (at your option) any later version.                                        *
 *                                                                            *
 * This program is distributed in the hope that it will be useful,            *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of             *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              *
 * GNU Lesser General Public License for more details.                        *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public License   *
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.      *
 * -------------------------------------------------------------------------- */

#include "amoebaCudaKernels.h"
#include "kCalculateAmoebaCudaUtilities.h"

static __constant__ cudaGmxSimulation cSim;
static __constant__ cudaAmoebaGmxSimulation cAmoebaSim;

void SetCalculateAmoebaCudaFixedEFieldSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaFixedEFieldSim: hipMemcpyToSymbol: SetSim copy to cSim failed");
    status         = hipMemcpyToSymbol(HIP_SYMBOL(cAmoebaSim), &amoebaGpu->amoebaSim, sizeof(cudaAmoebaGmxSimulation));    
    RTERROR(status, "SetCalculateAmoebaCudaFixedEFieldSim: hipMemcpyToSymbol: SetSim copy to cAmoebaSim failed");
}

void GetCalculateAmoebaCudaFixedEFieldSim(amoebaGpuContext amoebaGpu)
{
    hipError_t status;
    gpuContext gpu = amoebaGpu->gpuContext;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaGmxSimulation));    
    RTERROR(status, "GetCalculateAmoebaCudaFixedEFieldSim: hipMemcpyFromSymbol: SetSim copy from cSim failed");
    status = hipMemcpyFromSymbol(&amoebaGpu->amoebaSim, HIP_SYMBOL(cAmoebaSim), sizeof(cudaAmoebaGmxSimulation));         
    RTERROR(status, "GetCalculateAmoebaCudaFixedEFieldSim: hipMemcpyFromSymbol: SetSim copy from cAmoebaSim failed");
}

// reduce psWorkArray_3_1 -> EField
// reduce psWorkArray_3_2 -> EFieldPolar

static void kReduceE_Fields_kernel(amoebaGpuContext amoebaGpu )
{
    gpuContext gpu = amoebaGpu->gpuContext;
    kReduceFields_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.bsf_reduce_threads_per_block>>>(
                               gpu->sim.paddedNumberOfAtoms*3, gpu->sim.outputBuffers,
                               amoebaGpu->psWorkArray_3_1->_pDevData, amoebaGpu->psE_Field->_pDevData, 0 );
    LAUNCHERROR("kReduceE_Fields1");

    kReduceFields_kernel<<<gpu->sim.nonbond_blocks, gpu->sim.bsf_reduce_threads_per_block>>>(
                               gpu->sim.paddedNumberOfAtoms*3, gpu->sim.outputBuffers,
                               amoebaGpu->psWorkArray_3_2->_pDevData, amoebaGpu->psE_FieldPolar->_pDevData, 0 );
    LAUNCHERROR("kReduceE_Fields2");
}

// file includes FixedFieldParticle struct definition/load/unload struct and body kernel for fixed E-field

#undef GK
#include "kCalculateAmoebaCudaFixedFieldParticle.h"

// Include versions of the kernels for N^2 calculations.

#define METHOD_NAME(a, b) a##N2##b
#include "kCalculateAmoebaCudaFixedEField.h"
#define USE_OUTPUT_BUFFER_PER_WARP
#undef METHOD_NAME
#define METHOD_NAME(a, b) a##N2ByWarp##b
#include "kCalculateAmoebaCudaFixedEField.h"

/**---------------------------------------------------------------------------------------

   Compute fixed electric field

   @param amoebaGpu        amoebaGpu context
   @param gpu              OpenMM gpu Cuda context

   --------------------------------------------------------------------------------------- */

void cudaComputeAmoebaFixedEField( amoebaGpuContext amoebaGpu )
{
  
    gpuContext gpu    = amoebaGpu->gpuContext;

    kClearFields_3( amoebaGpu, 2 );

    static unsigned int threadsPerBlock = 0;
    if( threadsPerBlock == 0 ){ 
        unsigned int maxThreads;
        if (gpu->sm_version >= SM_20)
            maxThreads = 512; 
        else if (gpu->sm_version >= SM_12)
            maxThreads = 128; 
        else 
            maxThreads = 64;
        threadsPerBlock = std::min(getThreadsPerBlock(amoebaGpu, sizeof(FixedFieldParticle), gpu->sharedMemoryPerBlock ), maxThreads);
    }

    if (gpu->bOutputBufferPerWarp){
        kCalculateAmoebaFixedE_FieldN2ByWarpForces_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(FixedFieldParticle)*threadsPerBlock>>>(
                                                                           gpu->psWorkUnit->_pDevData,
                                                                           amoebaGpu->psWorkArray_3_1->_pDevData,
                                                                           amoebaGpu->psWorkArray_3_2->_pDevData );
    } else {

        kCalculateAmoebaFixedE_FieldN2Forces_kernel<<<gpu->sim.nonbond_blocks, threadsPerBlock, sizeof(FixedFieldParticle)*threadsPerBlock>>>(
                                                                           gpu->psWorkUnit->_pDevData,
                                                                           amoebaGpu->psWorkArray_3_1->_pDevData,
                                                                           amoebaGpu->psWorkArray_3_2->_pDevData );
    }

    LAUNCHERROR("kCalculateAmoebaFixedE_FieldN2Forces_kernel");
    kReduceE_Fields_kernel( amoebaGpu );
}
